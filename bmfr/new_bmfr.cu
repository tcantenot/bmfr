#include "hip/hip_runtime.h"
#include "new_bmfr.cuh"

// TODO
// - Try to pass everything as half (inputs and outputs)
//   --> #define WorldPositionType|WorldNormals|... float or half
// - LoadWorldPositions, LoadWorldNormals, ...
// - PixelCoordsToShiftedPixelCoords(pixelCoords, frameNumber, w, h) -> pixelCoords - BLOCK_EDGE_HALF + BLOCK_OFFSETS[frameNumber % BLOCK_OFFSETS_COUNT];
// - ShiftedPixelCoordsToPixelCoords(shiftedPixelCoords, frameNumber, w, h) -> shiftedPixelCoords + BLOCK_EDGE_HALF - BLOCK_OFFSETS[params.frameNumber % BLOCK_OFFSETS_COUNT];


#define K_SUPPORT_HALF16_ARITHMETIC (__CUDA_ARCH__ >= 530)

inline __device__ float Add(float lhs, float rhs) { return lhs + rhs; }
inline __device__ float Sub(float lhs, float rhs) { return lhs - rhs; }
inline __device__ float Mul(float lhs, float rhs) { return lhs * rhs; }
inline __device__ float Div(float lhs, float rhs) { return lhs / rhs; }

template <typename T>
struct Converter;

template <>
struct Converter<float>
{
	static inline __device__ float Convert(half x)  { return __half2float(x); }
	static inline __device__ float Convert(float x) { return x; }
};

template <>
struct Converter<half>
{
	static inline __device__ half Convert(half x)  { return x; }
	static inline __device__ half Convert(float x) { return __float2half(x); }
};

inline __device__ half Add(half lhs, half rhs)
{
	#if K_SUPPORT_HALF16_ARITHMETIC
	return __hadd(lhs, rhs);
	#else
	return __float2half(__half2float(lhs) + __half2float(rhs));
	#endif
}

inline __device__ void Add3(half const * lhs, half const * rhs, half * res)
{
	#if K_SUPPORT_HALF16_ARITHMETIC
	half2 tmp = __hadd2(__halves2half2(lhs[0], lhs[1]), __halves2half2(rhs[0], rhs[1]));
	res[0] = __low2half(tmp);
	res[1] = __high2half(tmp);
	res[2] = __hadd(lhs[2], rhs[2]);
	#else
	res[0] = __float2half(__half2float(lhs[0]) + __half2float(rhs[0]));
	res[1] = __float2half(__half2float(lhs[1]) + __half2float(rhs[1]));
	res[2] = __float2half(__half2float(lhs[2]) + __half2float(rhs[2]));
	#endif
}

inline __device__ void Add(half lhs[3], half rhs[3], half res[3])
{
	Add3(lhs, rhs, res);
}

inline __device__ half Sub(half lhs, half rhs)
{
	#if K_SUPPORT_HALF16_ARITHMETIC
	return __hsub(lhs, rhs);
	#else
	return __float2half(__half2float(lhs) - __half2float(rhs));
	#endif
}

inline __device__ void Sub3(half const * lhs, half const * rhs, half * res)
{
	#if K_SUPPORT_HALF16_ARITHMETIC
	half2 tmp = __hsub2(__halves2half2(lhs[0], lhs[1]), __halves2half2(rhs[0], rhs[1]));
	res[0] = __low2half(tmp);
	res[1] = __high2half(tmp);
	res[2] = __hsub(lhs[2], rhs[2]);
	#else
	res[0] = __float2half(__half2float(lhs[0]) - __half2float(rhs[0]));
	res[1] = __float2half(__half2float(lhs[1]) - __half2float(rhs[1]));
	res[2] = __float2half(__half2float(lhs[2]) - __half2float(rhs[2]));
	#endif
}

inline __device__ void Sub(half lhs[3], half rhs[3], half res[3])
{
	Sub3(lhs, rhs, res);
}

inline __device__ half Mul(half lhs, half rhs)
{
	#if K_SUPPORT_HALF16_ARITHMETIC
	return __hmul(lhs, rhs);
	#else
	return __float2half(__half2float(lhs) * __half2float(rhs));
	#endif
}

inline __device__ void Mul3(half const * lhs, half const * rhs, half * res)
{
	#if K_SUPPORT_HALF16_ARITHMETIC
	half2 tmp = __hmul2(__halves2half2(lhs[0], lhs[1]), __halves2half2(rhs[0], rhs[1]));
	res[0] = __low2half(tmp);
	res[1] = __high2half(tmp);
	res[2] = __hmul(lhs[2], rhs[2]);
	#else
	res[0] = __float2half(__half2float(lhs[0]) * __half2float(rhs[0]));
	res[1] = __float2half(__half2float(lhs[1]) * __half2float(rhs[1]));
	res[2] = __float2half(__half2float(lhs[2]) * __half2float(rhs[2]));
	#endif
}

inline __device__ void Mul(half lhs[3], half rhs[3], half res[3])
{
	Mul3(lhs, rhs, res);
}

inline __device__ half Div(half lhs, half rhs)
{
	#if K_SUPPORT_HALF16_ARITHMETIC
	return __hdiv(lhs, rhs);
	#else
	return __float2half(__half2float(lhs) / __half2float(rhs));
	#endif
}

inline __device__ void Div3(half const * lhs, half const * rhs, half * res)
{
	#if K_SUPPORT_HALF16_ARITHMETIC
	half2 tmp = __h2div(__halves2half2(lhs[0], lhs[1]), __halves2half2(rhs[0], rhs[1]));
	res[0] = __low2half(tmp);
	res[1] = __high2half(tmp);
	res[2] = __hdiv(lhs[2], rhs[2]);
	#else
	res[0] = __float2half(__half2float(lhs[0]) / __half2float(rhs[0]));
	res[1] = __float2half(__half2float(lhs[1]) / __half2float(rhs[1]));
	res[2] = __float2half(__half2float(lhs[2]) / __half2float(rhs[2]));
	#endif
}

inline __device__ void Div(half lhs[3], half rhs[3], half res[3])
{
	Div3(lhs, rhs, res);
}

inline __device__ half Sqrt(half x)
{
	return __float2half(sqrt(__half2float(x)));
}


template <>
struct tvec3<half> : public tcvec3<half>
{
	__device__ tvec3() { x = { 0 }; y = { 0 }; z = { 0 }; }
	__device__ explicit tvec3(half v) { x = y = z = v; }
	__device__ tvec3(half xx, half yy, half zz) { x = xx; y = yy; z = zz; }
	__device__ tvec3 & operator=(tvec3 const & o) { x = o.x; y = o.y; z = o.z; return *this; }
	__device__ tvec3 operator+(tvec3 const & o) const { tvec3 v; Add3(&x, &o.x, &v.x); return v; }
	__device__ tvec3 operator-(tvec3 const & o) const { tvec3 v; Sub3(&x, &o.x, &v.x); return v; }
	__device__ tvec3 operator*(tvec3 const & o) const { tvec3 v; Mul3(&x, &o.x, &v.x); return v; }
	__device__ tvec3 operator/(tvec3 const & o) const { tvec3 v; Div3(&x, &o.x, &v.x); return v; }

	__device__ tvec3 const & operator+=(tvec3 const & o) { Add3(&x, &o.x, &x); return *this; }
	__device__ tvec3 const & operator-=(tvec3 const & o) { Sub3(&x, &o.x, &x); return *this; }
	__device__ tvec3 const & operator*=(tvec3 const & o) { Mul3(&x, &o.x, &x); return *this; }
	__device__ tvec3 const & operator/=(tvec3 const & o) { Div3(&x, &o.x, &x); return *this; }

	__device__ tvec3 const & operator+=(half v) { half vv[3] = { v, v, v }; Add3(&x, vv, &x); return *this; }
	__device__ tvec3 const & operator-=(half v) { half vv[3] = { v, v, v }; Sub3(&x, vv, &x); return *this; }
	__device__ tvec3 const & operator*=(half v) { half vv[3] = { v, v, v }; Mul3(&x, vv, &x); return *this; }
	__device__ tvec3 const & operator/=(half v) { half vv[3] = { v, v, v }; Div3(&x, vv, &x); return *this; }
};

template <>
struct tvec4<half> : public tcvec4<half>
{
	__device__ tvec3<half> xyz() const { return tvec3<half>(x, y, z); }
};

using vec3h = tvec3<half>;
using vec4h = tvec4<half>;

inline __device__ half Lerp(half a, half b, half t)
{
	#if K_SUPPORT_HALF16_ARITHMETIC
	half one_minus_t = __hsub(__float2half(1.0f), t);
	half2 partial_sums = __hmul2(__halves2half2(one_minus_t, t), __halves2half2(a, b));
	return __hadd(__low2half(partial_sums), __high2half(partial_sums));
	#else
	float ft = __half2float(t);
	float fa = __half2float(a);
	float fb = __half2float(b);
	return __float2half((1.0f - ft) * fa + ft * fb);
	#endif
}

// TODO: benchmark
inline __device__ vec3h Lerp(vec3h const & a, vec3h const & b, vec3h const & t)
{
	#if 0
	half h1 = __float2half(1.0f);

	half2 one_minus_t_xy = __hsub2(__halves2half2(h1, t.x), __halves2half2(h1, t.y));
	half2 lhs_xy = __hmul2(one_minus_t_xy, __halves2half2(a.x, a.y));
	half2 rhs_xy = __hmul2(__halves2half2(t.x, t.y), __halves2half2(b.x, b.y));
	half2 res_xy = __hadd2(lhs_xy, rhs_xy);

	half one_minus_t_z = __hsub(h1, t.z);
	half2 partial_sums_z = __hmul2(__halves2half2(one_minus_t_z, t.z), __halves2half2(a.z, b.z));
	half res_z = __hadd(__low2half(partial_sums_z), __high2half(partial_sums_z));

	return vec3h(__low2half(res_xy), __high2half(res_xy), res_z);
	#else
	return vec3h(Lerp(a.x, b.x, t.x), Lerp(a.y, b.y, t.y), Lerp(a.z, b.z, t.z));
	#endif
}

// TODO: benchmark
inline __device__ vec3h Lerp(vec3h const & a, vec3h const & b, half t)
{
	#if 0
	half one_minus_t = __hsub(__float2half(1.0f), t);

	half2 lhs_xy = __hmul2(__halves2half2(one_minus_t, one_minus_t), __halves2half2(a.x, a.y));
	half2 rhs_xy = __hmul2(__halves2half2(t, t), __halves2half2(b.x, b.y));
	half2 res_xy = __hadd2(lhs_xy, rhs_xy);

	half2 partial_sums_z = __hmul2(__halves2half2(one_minus_t, t), __halves2half2(a.z, b.z));
	half res_z = __hadd(__low2half(partial_sums_z), __high2half(partial_sums_z));

	return vec3h(__low2half(res_xy), __high2half(res_xy), res_z);
	#else
	return vec3h(Lerp(a.x, b.x, t), Lerp(a.y, b.y, t), Lerp(a.z, b.z, t));
	#endif
}


template <typename In, typename Out>
struct LoadStoreHelper;

template <>
struct LoadStoreHelper<float, float>
{
	static inline __device__ vec3 load3(float const * K_RESTRICT buffer, unsigned int index)
	{
		#if OPTIMIZE_LOAD_STORE
		return (*reinterpret_cast<vec4 const *>(buffer + index * 3)).xyz();
		#else
		return vec3(buffer[index * 3 + 0], buffer[index * 3 + 1], buffer[index * 3 + 2]);
		#endif
	}

	static inline __device__ void store3(float * K_RESTRICT buffer, unsigned int index, vec3 const & value)
	{
		#if OPTIMIZE_LOAD_STORE
		*reinterpret_cast<vec3 *>(buffer + index * 3) = value;
		#else
		buffer[index * 3 + 0] = value.x;
		buffer[index * 3 + 1] = value.y;
		buffer[index * 3 + 2] = value.z;
		#endif
	}
};

template <>
struct LoadStoreHelper<half, half>
{
	static inline __device__ vec3h load3(half const * K_RESTRICT buffer, unsigned int index)
	{
		#if OPTIMIZE_LOAD_STORE
		return (*reinterpret_cast<vec4h const *>(buffer + index * 3)).xyz();
		#else
		return vec3h(buffer[index * 3 + 0], buffer[index * 3 + 1], buffer[index * 3 + 2]);
		#endif
	}

	static inline __device__ void store3(half * K_RESTRICT buffer, unsigned int index, vec3h const & value)
	{
		#if OPTIMIZE_LOAD_STORE
		*reinterpret_cast<vec3h *>(buffer + index * 3) = value;
		#else
		buffer[index * 3 + 0] = value.x;
		buffer[index * 3 + 1] = value.y;
		buffer[index * 3 + 2] = value.z;
		#endif
	}
};

template <>
struct LoadStoreHelper<float, half>
{
	static inline __device__ vec3h load3(float const * K_RESTRICT buffer, unsigned int index)
	{
		#if OPTIMIZE_LOAD_STORE
		vec4 const * v = reinterpret_cast<vec4 const *>(buffer + index * 3);
		return vec3h(__float2half(v->x), __float2half(v->y), __float2half(v->z));
		#else
		return vec3h(__float2half(buffer[index * 3 + 0]),__float2half(buffer[index * 3 + 1]), __float2half(buffer[index * 3 + 2]));
		#endif
	}

	static inline __device__ void store3(half * K_RESTRICT buffer, unsigned int index, vec3 const & value)
	{
		#if OPTIMIZE_LOAD_STORE
		vec3h * v = reinterpret_cast<vec3h *>(buffer + index * 3);
		v->x = __float2half(value.x);
		v->y = __float2half(value.y);
		v->z = __float2half(value.z);
		#else
		buffer[index * 3 + 0] = __float2half(value.x);
		buffer[index * 3 + 1] = __float2half(value.y);
		buffer[index * 3 + 2] = __float2half(value.z);
		#endif
	}
};

template <>
struct LoadStoreHelper<half, float>
{
	static inline __device__ vec3 load3(half const * K_RESTRICT buffer, unsigned int index)
	{
		#if OPTIMIZE_LOAD_STORE
		vec4h const * v = reinterpret_cast<vec4h const *>(buffer + index * 3);
		return vec3(__half2float(v->x), __half2float(v->y), __half2float(v->z));
		#else
		return vec3(__half2float(buffer[index * 3 + 0]),__half2float(buffer[index * 3 + 1]), __half2float(buffer[index * 3 + 2]));
		#endif
	}

	static inline __device__ void store3(float * K_RESTRICT buffer, unsigned int index, vec3h const & value)
	{
		#if OPTIMIZE_LOAD_STORE
		vec3 * v = reinterpret_cast<vec3 *>(buffer + index * 3);
		v->x = __half2float(value.x);
		v->y = __half2float(value.y);
		v->z = __half2float(value.z);
		#else
		buffer[index * 3 + 0] = __half2float(value.x);
		buffer[index * 3 + 1] = __half2float(value.y);
		buffer[index * 3 + 2] = __half2float(value.z);
		#endif
	}
};


template <typename Out, typename In>
inline __device__ tvec3<Out> load3(In const * K_RESTRICT buffer, unsigned int index)
{
	return LoadStoreHelper<In, Out>::load3(buffer, index);
}

template <typename Out, typename In>
inline __device__ void store3(Out * K_RESTRICT buffer, unsigned int index, tvec3<In> const & value)
{
	LoadStoreHelper<In, Out>::store3(buffer, index, value);
}

inline __device__ void store_feature(half * buffer, unsigned int index, half value)
{
	buffer[index] = value;
}

// Compute features ////////////////////////////////////////////////////////////

template <typename T, typename U, typename FeatureType>
inline __device__ void compute_features_without_color(
	tvec3<T> const & world_position,
	tvec3<U> const & normal,
	FeatureType * features
)
{
	features[0]  = Converter<FeatureType>::Convert(1.0f);
	features[1]  = Converter<FeatureType>::Convert(normal.x);
	features[2]  = Converter<FeatureType>::Convert(normal.y);
	features[3]  = Converter<FeatureType>::Convert(normal.z);
	features[4]  = Converter<FeatureType>::Convert(world_position.x);
	features[5]  = Converter<FeatureType>::Convert(world_position.y);
	features[6]  = Converter<FeatureType>::Convert(world_position.z);
	features[7]  = Converter<FeatureType>::Convert(Mul(world_position.x, world_position.x));
	features[8]  = Converter<FeatureType>::Convert(Mul(world_position.y, world_position.y));
	features[9]  = Converter<FeatureType>::Convert(Mul(world_position.z, world_position.z));
}

template <typename T, typename U, typename V, typename FeatureType>
inline __device__ void compute_features(
	tvec3<T> const & world_position,
	tvec3<U> const & normal,
	tvec3<V> const & noisy_1spp_color,
	FeatureType features[BUFFER_COUNT]
)
{
	compute_features_without_color(world_position, normal, features);
	features[10] = Converter<FeatureType>::Convert(noisy_1spp_color.x);
	features[11] = Converter<FeatureType>::Convert(noisy_1spp_color.y);
	features[12] = Converter<FeatureType>::Convert(noisy_1spp_color.z);
}

// Rescale features ////////////////////////////////////////////////////////////

inline __device__ void parallel_reduction_min_1024(
	float * K_RESTRICT result,
	float * K_RESTRICT pr_data_1024,
	const unsigned int index
)
{
	if(index < 256)
	{
		pr_data_1024[index] = Min(
			Min(pr_data_1024[index], pr_data_1024[index + 256]),
			Min(pr_data_1024[index + 512], pr_data_1024[index + 768])
		);
	}

	SyncThreads();

	parallel_reduction_min_256(result, pr_data_1024);
}

inline __device__ void parallel_reduction_max_1024(
	float * K_RESTRICT result,
	float * K_RESTRICT pr_data_1024,
	const unsigned int index
)
{
	if(index < 256)
	{
		pr_data_1024[index] = Max(
			Max(pr_data_1024[index], pr_data_1024[index + 256]),
			Max(pr_data_1024[index + 512], pr_data_1024[index + 768])
		);
	}

	SyncThreads();

	parallel_reduction_max_256(result, pr_data_1024);
}

inline __device__ void parallel_reduction_sum_1024(
	float * K_RESTRICT result,
	float * K_RESTRICT pr_data_1024,
	const unsigned int index
)
{
	if(index < 256)
		pr_data_1024[index] += pr_data_1024[index + 256] + pr_data_1024[index + 512] + pr_data_1024[index + 768];
	SyncThreads();

	parallel_reduction_sum_256(result, pr_data_1024, 0);
}

__global__ void rescale_world_positions_pr(
	RescaleFeaturesParams params,
	float const * world_positions,
	float * normalized_world_positions
)
{
	__shared__ float lds[1024];
	__shared__ float block_min;
	__shared__ float block_max;

	const ivec2 gtid = ivec2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	const int w = params.sizeX;
	const int h = params.sizeY;

	// Mirror indexed of the input. x and y are always less than one size out of
	// bounds if image dimensions are bigger than BLOCK_EDGE_LENGTH
	// BLOCK_EDGE_HALF = half block size (32/2 -> 16)
	const ivec2 pixel_without_mirror = gtid - BLOCK_EDGE_HALF + BLOCK_OFFSETS[params.frameNumber % BLOCK_OFFSETS_COUNT];

	// Pixel coordinates in [0, w-1]x[0, h-1]
	const ivec2 pixel = mirror2(pixel_without_mirror, ivec2(w, h));

	// Linear pixel index in image in [0, w*h-1]
	const int linear_pixel = pixel.y * w + pixel.x;

	// Current frame world position
	const vec3 v = load3<float>(world_positions, linear_pixel);

	// Note: assume group of size 1024
	const unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
		
	lds[tid] = v.x;
	SyncThreads();
	parallel_reduction_min_1024(&block_min, lds, tid);
	lds[tid] = v.x;
	SyncThreads();
	parallel_reduction_max_1024(&block_max, lds, tid);
	float scaledX = -Min(-scale(v.x, block_min, block_max), 0.0f); // Remove NaN

	lds[tid] = v.y;
	SyncThreads();
	parallel_reduction_min_1024(&block_min, lds, tid);
	lds[tid] = v.y;
	SyncThreads();
	parallel_reduction_max_1024(&block_max, lds, tid);
	float scaledY = -Min(-scale(v.y, block_min, block_max), 0.0f);

	lds[tid] = v.z;
	SyncThreads();
	parallel_reduction_min_1024(&block_min, lds, tid);
	lds[tid] = v.z;
	SyncThreads();
	parallel_reduction_max_1024(&block_max, lds, tid);
	float scaledZ = -Min(-scale(v.z, block_min, block_max), 0.0f);

	if(pixel_without_mirror.x >= 0 && pixel_without_mirror.x < w &&
	   pixel_without_mirror.y >= 0 && pixel_without_mirror.y < h
	)
	{
		//scaledX = Clamp(v.x / 15.0f, 0.f, 1.f);
		//scaledY = Clamp(v.y / 15.0f, 0.f, 1.f);
		//scaledZ = Clamp(v.z / 15.0f, 0.f, 1.f);
		store3(normalized_world_positions, linear_pixel, vec3(scaledX, scaledY, scaledZ));
	}
}

extern "C" void run_rescale_world_positions_pr(
	dim3 const & grid_size,
	dim3 const & block_size,
	RescaleFeaturesParams const & params,
	float const * world_positions,
	float * normalized_world_positions
)
{
	rescale_world_positions_pr<<<grid_size, block_size>>>(
		params,
		world_positions,
		normalized_world_positions
	);
}

// Accumulate noisy 1spp color kernel //////////////////////////////////////////

template <typename NormalType, typename PosType, typename InColorType, typename OutColorType, typename FeaturesType>
__device__ void template_accumulate_noisy_data_frame0(
	AccumulateNoisyDataKernelParams params,
	const NormalType * K_RESTRICT frame_normals,			// [in]  Frame (world) normals
	const PosType * K_RESTRICT frame_normalized_positions,	// [in]  Frame normalized world positions
	const InColorType * K_RESTRICT frame_noisy_1spp,		// [in]  Frame noisy 1spp color buffer
	OutColorType * K_RESTRICT frame_acc_noisy,				// [out] Frame accumulated noisy color
	unsigned char * K_RESTRICT frame_acc_num_spp,			// [out] Frame accumulated number of samples (for CMA)
	FeaturesType * K_RESTRICT features_data					// [out] Features buffer
)
{
	const ivec2 gtid = ivec2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	const int w = params.sizeX;
	const int h = params.sizeY;

	// Mirror indexed of the input. x and y are always less than one size out of
	// bounds if image dimensions are bigger than BLOCK_EDGE_LENGTH
	// BLOCK_EDGE_HALF = half block size (32/2 -> 16)
	const ivec2 pixel_without_mirror = gtid - BLOCK_EDGE_HALF + BLOCK_OFFSETS[params.frameNumber % BLOCK_OFFSETS_COUNT];

	// Pixel coordinates in [0, w-1]x[0, h-1]
	const ivec2 pixel = mirror2(pixel_without_mirror, ivec2(w, h));

	// Linear pixel index in image in [0, w*h-1]
	const int linear_pixel = pixel.y * w + pixel.x;
   
	// Current frame noisy color (1spp)
	// [Section 3.1]
	// The input for the real-time reconstruction filter is a 1 spp path-traced frame and its accompanying feature buffers.
	// The 1 spp frames are generated by using a rasterizer for producing the primary rays and feature buffers.
	// We use mipmapped textures in albedo. Next, we do so-called next event estimation: we trace one shadow ray towards
	// a random point in one random light source and then continue path tracing by sending one secondary ray to a random direction.
	// Namely, we use multiple importance sampling [Veach and Guibas 1995].
	// The direction of the secondary ray is decided based on importance sampling. We also trace a second shadow ray from the
	// intersection point of the secondary ray.
	// Consequently, the 1spp pixel input has one rasterized primary ray (non-noisy), one ray-traced secondary ray and two ray-traced shadow rays.
	const tvec3<InColorType> current_color = load3<InColorType>(frame_noisy_1spp, linear_pixel);

	// Current frame normalized world position ([0, 1])
	const tvec3<PosType> normalized_world_position = load3<PosType>(frame_normalized_positions, linear_pixel);

	// Current frame (world) normal
	const tvec3<NormalType> normal = load3<NormalType>(frame_normals, linear_pixel);

	// Compute the set of feature buffers used in the fitting
	FeaturesType features[BUFFER_COUNT];
	compute_features(normalized_world_position, normal, current_color, features);

	const unsigned int x_block = gtid.x / BLOCK_EDGE_LENGTH; // Block coordinate x
	const unsigned int y_block = gtid.y / BLOCK_EDGE_LENGTH; // Block coordinate y
	const unsigned int x_in_block = gtid.x % BLOCK_EDGE_LENGTH; // Thread coordinate x inside block in [0, BLOCK_EDGE_LENGTH-1]
	const unsigned int y_in_block = gtid.y % BLOCK_EDGE_LENGTH; // Thread coordinate y inside block in [0, BLOCK_EDGE_LENGTH-1]

	const unsigned int features_base_offset = x_in_block + y_in_block * BLOCK_EDGE_LENGTH +
		x_block * BLOCK_PIXELS * BUFFER_COUNT +
		y_block * params.worksetWithMarginBlockCountX *
		BLOCK_PIXELS * BUFFER_COUNT;
	
	// TODO: change layout of features buffer to allow 128-bit loads?
	// --> | Block 0 thread 0 feature 0 | Block 0 thread 0 feature 1 | ... | Block 0 thread 1 feature 0 | ... | Block N thread 0 feature 0 | ... | Block N thread T feature M |
	for(unsigned int featureIndex = 0; featureIndex < BUFFER_COUNT; ++featureIndex)
	{
		// Offset in feature buffer (data are concatenated)
		// | Block 0 thread 0 feature 0 | Block 0 thread 1 feature 0 | ... | Block 0 thread 0 feature M | ... | Block 1 thread 0 feature 0 | ... | Block N thread 0 feature 0 | ... | Block N thread T feature M |
		const unsigned int featureOffset = features_base_offset + featureIndex * BLOCK_PIXELS;
		store_feature(features_data, featureOffset, features[featureIndex]);
	}

	// The kernel works on a workset of size WORKSET_WITH_MARGINS_WIDTH x WORKSET_WITH_MARGINS_HEIGHT
	// -> the extra block margin is used to handle the offsets applied to reduce the block artifacts.
	// [Section 3.5]: "To aid the reduction of blockiness, BMFR processes each frame over a grid of non-overlapping
	//	blocks which is displaced with random offsets. These offsets prevent the artifacts that would arise from reusing
	// same block positions on a static scene with a static camera."
	// --> Only the pixels inside the image (after applying the offsets) should write to the output data that
	// have the same size of the input image
	if(pixel_without_mirror.x >= 0 && pixel_without_mirror.x < w &&
	   pixel_without_mirror.y >= 0 && pixel_without_mirror.y < h
	)
	{
		store3(frame_acc_noisy, linear_pixel, current_color); // Accumulated noisy color
		frame_acc_num_spp[linear_pixel] = 1; // Store current number of samples accumulated (for CMA)
	}
}

__global__ void accumulate_noisy_data_frame0(
	AccumulateNoisyDataKernelParams params,
	const float * K_RESTRICT frame_normals,					// [in]  Frame (world) normals
	const float * K_RESTRICT frame_normalized_positions,	// [in]  Frame normalized world positions
	const float * K_RESTRICT frame_noisy_1spp,				// [in]  Frame noisy 1spp color buffer
		  float * K_RESTRICT frame_acc_noisy,				// [out] Frame accumulated noisy color
		  unsigned char * K_RESTRICT frame_acc_num_spp,		// [out] Frame accumulated number of samples (for CMA)
	#if USE_HALF_PRECISION_IN_FEATURES_DATA
	half * K_RESTRICT features_data							// [out] Features buffer (half-precision)
	#else
	float * K_RESTRICT features_data						// [out] Features buffer (single-precision)
	#endif
)
{
	template_accumulate_noisy_data_frame0(
		params,
		frame_normals,
		frame_normalized_positions,
		frame_noisy_1spp,
		frame_acc_noisy,
		frame_acc_num_spp,
		features_data
	);
}

__global__ void accumulate_noisy_data_frame0_16bits(
	AccumulateNoisyDataKernelParams params,
	const half * K_RESTRICT frame_normals,				// [in]  Frame (world) normals
	const half * K_RESTRICT frame_normalized_positions,	// [in]  Frame normalized world positions
	const half * K_RESTRICT frame_noisy_1spp,			// [in]  Frame noisy 1spp color buffer
	half * K_RESTRICT frame_acc_noisy,					// [out] Frame accumulated noisy color
	unsigned char * K_RESTRICT frame_acc_num_spp,		// [out] Frame accumulated number of samples (for CMA)
	half * K_RESTRICT features_data						// [out] Features buffer (half-precision)
)
{
	template_accumulate_noisy_data_frame0(
		params,
		frame_normals,
		frame_normalized_positions,
		frame_noisy_1spp,
		frame_acc_noisy,
		frame_acc_num_spp,
		features_data
	);
}

extern "C" void run_accumulate_noisy_data_frame0(
	dim3 const & grid_size,
	dim3 const & block_size,
	AccumulateNoisyDataKernelParams params,
	const float * K_RESTRICT frame_normals,					// [in]  Frame (world) normals
	const float * K_RESTRICT frame_normalized_positions,	// [in]  Frame normalized world positions
	const float * K_RESTRICT frame_noisy_1spp,				// [in]  Frame noisy 1spp color buffer
		  float * K_RESTRICT frame_acc_noisy,				// [out] Accumulated noisy color
		  unsigned char * K_RESTRICT frame_acc_num_spp,		// [out] Accumulated number of samples (for CMA)
	#if USE_HALF_PRECISION_IN_FEATURES_DATA
	half * K_RESTRICT features_data							// [out] Features buffer (half-precision)
	#else
	float * K_RESTRICT features_data						// [out] Features buffer (single-precision)
	#endif
)
{
	accumulate_noisy_data_frame0<<<grid_size, block_size>>>(
		params,
		frame_normals,
		frame_normalized_positions,
		frame_noisy_1spp,
		frame_acc_noisy,
		frame_acc_num_spp,
		features_data
	);
}

template <typename NormalType, typename PosType, typename InColorType, typename OutColorType, typename FeaturesType>
__device__ void template_accumulate_noisy_data(
	AccumulateNoisyDataKernelParams params,
	vec2 * K_RESTRICT out_prev_frame_pixel,					// [out] Previous frame pixel coordinates (after reprojection)
	unsigned char* K_RESTRICT accept_bools,					// [out] Validity mask of bilinear samples in previous frame (after reprojection)
	const NormalType * K_RESTRICT frame_normals,			// [in]  Current  frame (world) normals
	const NormalType * K_RESTRICT prev_frame_normals,		// [in]  Previous frame (world) normals
	const PosType * K_RESTRICT frame_positions,				// [in]  Current  frame world positions
	const PosType * K_RESTRICT prev_frame_positions,		// [in]  Previous frame world positions
	const PosType * K_RESTRICT frame_normalized_positions,	// [in]  Frame normalized world positions
	const InColorType * K_RESTRICT frame_noisy_1spp,		// [in]  Frame noisy 1spp color
		  OutColorType * K_RESTRICT frame_acc_noisy,		// [out] Current  frame accumulated noisy color
	const OutColorType * K_RESTRICT prev_frame_acc_noisy,	// [in]  Previous frame accumulated noisy color
	const unsigned char * K_RESTRICT prev_frame_acc_spp,	// [in]  Previous frame accumulated number of samples (for CMA)
		  unsigned char * K_RESTRICT frame_acc_num_spp,		// [out] Current  frame accumulated number of samples (for CMA)
	FeaturesType * K_RESTRICT features_data,				// [out] Features buffer
	const mat4x4 prev_frame_camera_matrix,					// [in]  ViewProj matrix of previous frame
	const vec2 pixel_offset
)
{
	const ivec2 gtid = ivec2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	const int w = params.sizeX;
	const int h = params.sizeY;

	// Mirror indexed of the input. x and y are always less than one size out of
	// bounds if image dimensions are bigger than BLOCK_EDGE_LENGTH
	// BLOCK_EDGE_HALF = half block size (32/2 -> 16)
	const ivec2 pixel_without_mirror = gtid - BLOCK_EDGE_HALF + BLOCK_OFFSETS[params.frameNumber % BLOCK_OFFSETS_COUNT];

	// Pixel coordinates in [0, w-1]x[0, h-1]
	const ivec2 pixel = mirror2(pixel_without_mirror, ivec2(w, h));

	// Linear pixel index in image in [0, w*h-1]
	const int linear_pixel = pixel.y * w + pixel.x;
   
	// Current frame noisy color (1spp)
	// [Section 3.1]
	// The input for the real-time reconstruction filter is a 1 spp path-traced frame and its accompanying feature buffers.
	// The 1 spp frames are generated by using a rasterizer for producing the primary rays and feature buffers.
	// We use mipmapped textures in albedo. Next, we do so-called next event estimation: we trace one shadow ray towards
	// a random point in one random light source and then continue path tracing by sending one secondary ray to a random direction.
	// Namely, we use multiple importance sampling [Veach and Guibas 1995].
	// The direction of the secondary ray is decided based on importance sampling. We also trace a second shadow ray from the
	// intersection point of the secondary ray.
	// Consequently, the 1spp pixel input has one rasterized primary ray (non-noisy), one ray-traced secondary ray and two ray-traced shadow rays.
	const vec3 current_color = load3<float>(frame_noisy_1spp, linear_pixel);

	// Current frame normalized world position ([0, 1])
	const tvec3<PosType> normalized_world_position = load3<PosType>(frame_normalized_positions, linear_pixel);

	// Current frame (world) normal
	const vec3 normal = load3<float>(frame_normals, linear_pixel);

	// Project current world position into previous frame with the previous ViewProj matrix

	// Current frame world position
	// TODO: instead of comparing full world positions, use only depth of current, previous and
	// reprojected frame to detect (dis)occlusion
	const vec4 world_position = vec4(load3<float>(frame_positions, linear_pixel), 1.f);

	// Matrix multiplication and normalization to 0..1
	vec2 prev_frame_uv;

	// TODO: send matrix transposed
	prev_frame_uv.x = Dot(prev_frame_camera_matrix.row(0), world_position); // Transform x
	prev_frame_uv.y = Dot(prev_frame_camera_matrix.row(1), world_position); // Transform y
	// No need for z-buffer in accumulation of the noisy data
	// -> might be useful if we use it to detect disocclusion
	// --> compare previous z (store previous frame Z-buffer) with prev_frame_pixel_uv.z
	//prev_frame_uv.z = Dot(prev_frame_camera_matrix.row(2), world_position); // Transform z
	prev_frame_uv /= Dot(prev_frame_camera_matrix.row(3), world_position);

	prev_frame_uv = prev_frame_uv * vec2(0.5f) + vec2(0.5f);

	// Compute the pixel coordinates in the previous frame (in [0, w-1]x[0, h-1])
	vec2 prev_frame_pixel_f = prev_frame_uv * vec2(w, h);

	// Apply offset (TODO: what offset??? seems to always be 0.5... Maybe TAA/ray subpixel offsets -> send 1-pixel_offset.y)
	// TODO: try to remove this
	prev_frame_pixel_f -= vec2(pixel_offset.x, 1 - pixel_offset.y);

	// Convert into integer pixel coordinates (round down)
	const ivec2 prev_frame_pixel_i = FloatToIntRd(prev_frame_pixel_f);

	// Compute bilinear weights (for bilinear sampling)
	// TODO: implement bicubic Catmull-Rom (for sharpness)? => would need to perform more fetches and store more "validity bits" in mask
	const ivec2 offsets[4] = { ivec2(0, 0), ivec2(1, 0), ivec2(0, 1), ivec2(1, 1) };

	const vec2 prev_pixel_fract = prev_frame_pixel_f - vec2(prev_frame_pixel_i);
	const vec2 one_minus_prev_pixel_fract = 1.f - prev_pixel_fract;

	float weights[4];
	weights[0] = one_minus_prev_pixel_fract.x * one_minus_prev_pixel_fract.y;
	weights[1] = prev_pixel_fract.x           * one_minus_prev_pixel_fract.y;
	weights[2] = one_minus_prev_pixel_fract.x * prev_pixel_fract.y;
	weights[3] = prev_pixel_fract.x           * prev_pixel_fract.y;
	float total_weight = 0.f;

	// Bit mask telling which previous frame (bilinear) samples are valid under reprojection into current frame
	unsigned char store_accept = 0x00;
	vec3 previous_color = vec3(0.f, 0.f, 0.f);
	float sample_spp = 0.f;

	// Bilinear sampling
	for(int i = 0; i < 4; ++i)
	{
		ivec2 sample_location = prev_frame_pixel_i + offsets[i];

		// Check if previous frame color can be used based on its screen location
		if(sample_location.x >= 0 && sample_location.y >= 0 &&
			sample_location.x < w  && sample_location.y < h
		)
		{
			const int linear_sample_location = sample_location.y * w + sample_location.x;

			// Fetch previous frame world position
			vec3 prev_world_position = load3<float>(prev_frame_positions, linear_sample_location);

			// TODO: find a another metric to discard wrong history
			// -> world position is normalized to [0, 1]...
			// OR bind both normalized and non-normalized
			// Compute world distance squared
			vec3 position_difference = prev_world_position - world_position.xyz();
			float position_distance_squared = Dot(position_difference, position_difference);

			// World position distance discard
			if(position_distance_squared < float(POSITION_LIMIT_SQUARED))
			{
				// Fetch previous frame normal
				vec3 prev_normal = load3<float>(prev_frame_normals, linear_sample_location);

				// Distance of the normals
				// TODO: could use some other distance metric (e.g. angle), but we use hard
				// experimentally found threshold -> means that the metric doesn't matter.
				vec3 normal_difference = prev_normal - normal;
				float normal_distance_squared = Dot(normal_difference, normal_difference);

				if(normal_distance_squared < float(NORMAL_LIMIT_SQUARED))
				{
					// Pixel passes all tests so store it to "validity bitmask"
					store_accept |= 1 << i;

					// Accumulate number of samples
					sample_spp += weights[i] * float(prev_frame_acc_spp[linear_sample_location]);

					// Accumulate previous noisy 1spp color
					previous_color += weights[i] * load3<float>(prev_frame_acc_noisy, linear_sample_location);

					// Acumulate weights
					total_weight += weights[i];
				}
			}
		}
	}

	// Blending factor with history buffer
	// Blend_alpha 1.f means that only current frame color is used. The value is changed if sample from previous frame can be used
	float blend_alpha = 1.f;
	if(total_weight > 0.f)
	{
		previous_color /= total_weight;
		sample_spp /= total_weight;

		// Cumulative Moving Average (CMA)
		// CMA_n = (x_1 + x_2 + ... + x_n) / n
		// <=> (x_1 + x_2 + ... + x_n) = n * CMA_n
		// CMA_(n+1) = (x_1 + x_2 + ... + x_n + x_(n+1)) / (n + 1)
		//			  = (n * CMA_n + x_(n+1)) / (n + 1)
		//			  = n/(n+1) * CMA_n + 1/(n+1) * x_(n+1)
		//			  = (n+1-1)/(n+1) * CMA_n + 1/(n+1) * x_(n+1)
		//			  = (1 - 1/(n+1)) * CMA_n + 1/(n+1) * x_(n+1)
		//			  = lerp(CMA_n, x_(n+1), 1/(n+1))
		blend_alpha = 1.f / (sample_spp + 1.f);

		// Blend_alpha is dymically decided so that the result is average
		// of all samples (cumulative moving average) until the cap defined by
		// BLEND_ALPHA is reached (exponential moving average: EMA_(n+1) = (1 - a) * EMA_n + a * x_(n+1) = lerp(EMA_n, x_(n+1), a))

		// [Section 3.2]
		// We start by computing a cumulative moving average of the samples, 
		// and use the exponential moving average only after the cumulative moving average weight
		// of the new sample would be less than BLEND_ALPHA (e.g 20%).
		// The use of regular average on the first frames and after occlusions makes sure that
		// the first samples do not get an excessively high weight, and limiting the weight to a minimum
		// of BLEND_APLHA (e.g 20%) makes sure that the aged data fades away.
		blend_alpha = Max(blend_alpha, BLEND_ALPHA);
	}

	// Store new spp
	unsigned char new_spp = 1;
	if(blend_alpha < 1.f) // alpha = 1.f means we ignore history
	{
		// Note: we accumulate at most 255 samples for the cumulative moving average (which is more than enough because of
		// the threshold BLEND_ALPHA that switch to exponential moving average).
		// E.g: BLEND_ALPHA = 0.2 = 1 / (n + 1) <=> n = (1 - 0.2) / 0.2 = 4 => above 4 samples for a pixel, we switch to
		// exponential moving average with alpha = 20%
		// max n = 255 <=> min BLEND_ALPHA = 1.0 / (255 + 1) = 0.0039
		
		// TODO: store "validity mask" along with the "spp" in 8 bits: 4-bit validity mask | 4-bit spp
		// 4-bit spp <=> max n = 2^4-1 = 15 <=> min BLEND_ALPHA = 1.0 / (5 + 1) = 0.0625

		new_spp = (sample_spp > 254.f) ? 255 : convert_uchar_sat_rte(sample_spp) + 1;
	}

	vec3 new_color = Lerp(previous_color, current_color, blend_alpha);

	// Compute the set of feature buffers used in the fitting
	FeaturesType features[BUFFER_COUNT];
	compute_features(normalized_world_position, normal, current_color, features);

	const unsigned int x_block = gtid.x / BLOCK_EDGE_LENGTH; // Block coordinate x
	const unsigned int y_block = gtid.y / BLOCK_EDGE_LENGTH; // Block coordinate y
	const unsigned int x_in_block = gtid.x % BLOCK_EDGE_LENGTH; // Thread coordinate x inside block in [0, BLOCK_EDGE_LENGTH-1]
	const unsigned int y_in_block = gtid.y % BLOCK_EDGE_LENGTH; // Thread coordinate y inside block in [0, BLOCK_EDGE_LENGTH-1]

	const unsigned int features_base_offset = x_in_block + y_in_block * BLOCK_EDGE_LENGTH +
		x_block * BLOCK_PIXELS * BUFFER_COUNT +
		y_block * params.worksetWithMarginBlockCountX *
		BLOCK_PIXELS * BUFFER_COUNT;
	
	// TODO: change layout of features buffer to allow 128-bit loads?
	// --> | Block 0 thread 0 feature 0 | Block 0 thread 0 feature 1 | ... | Block 0 thread 1 feature 0 | ... | Block N thread 0 feature 0 | ... | Block N thread T feature M |
	for(unsigned int featureIndex = 0; featureIndex < BUFFER_COUNT; ++featureIndex)
	{
		// Offset in feature buffer (data are concatenated)
		// | Block 0 thread 0 feature 0 | Block 0 thread 1 feature 0 | ... | Block 0 thread 0 feature M | ... | Block 1 thread 0 feature 0 | ... | Block N thread 0 feature 0 | ... | Block N thread T feature M |
		const unsigned int featureOffset = features_base_offset + featureIndex * BLOCK_PIXELS;
		store_feature(features_data, featureOffset, features[featureIndex]);
	}

	// The kernel works on a workset of size WORKSET_WITH_MARGINS_WIDTH x WORKSET_WITH_MARGINS_HEIGHT
	// -> the extra block margin is used to handle the offsets applied to reduce the block artifacts.
	// [Section 3.5]: "To aid the reduction of blockiness, BMFR processes each frame over a grid of non-overlapping
	//	blocks which is displaced with random offsets. These offsets prevent the artifacts that would arise from reusing
	// same block positions on a static scene with a static camera."
	// --> Only the pixels inside the image (after applying the offsets) should write to the output data that
	// have the same size of the input image
	if(pixel_without_mirror.x >= 0 && pixel_without_mirror.x < w &&
	   pixel_without_mirror.y >= 0 && pixel_without_mirror.y < h
	)
	{
		store3(frame_acc_noisy, linear_pixel, new_color); // Accumulated noisy 1spp
		out_prev_frame_pixel[linear_pixel] = prev_frame_pixel_f; // Previous frame pixel coordinates (to sample history)
		accept_bools[linear_pixel] = store_accept; // "Previous frame bilinear samples validity" bitmask
		frame_acc_num_spp[linear_pixel] = new_spp; // Store current number of samples accumulated (for CMA)

		// Kernel debug: stored in acc_noisy buffer
		#if 0
		vec3 debug = vec3(0.0f);
		//debug = vec3(prev_frame_uv.x, prev_frame_uv.y, 0);
		//debug = vec3(blend_alpha);
		debug = HeatMap(Saturate(float(new_spp) / 255.f));
		//debug = vec3(float(store_accept > 0));
		//debug = vec3(float(store_accept == ((1 << 4) - 1)));
		store3(acc_noisy, linear_pixel, debug);
		#endif
	}
}


__global__ void new_accumulate_noisy_data(
	AccumulateNoisyDataKernelParams params,
	vec2 * K_RESTRICT out_prev_frame_pixel,					// [out] Previous frame pixel coordinates (after reprojection)
	unsigned char* K_RESTRICT accept_bools,					// [out] Validity mask of bilinear samples in previous frame (after reprojection)
	const float * K_RESTRICT frame_normals,					// [in]  Current  frame (world) normals
	const float * K_RESTRICT prev_frame_normals,			// [in]  Previous frame (world) normals
	const float * K_RESTRICT frame_positions,				// [in]  Current  frame world positions
	const float * K_RESTRICT prev_frame_positions,			// [in]  Previous frame world positions
	const float * K_RESTRICT frame_normalized_positions,	// [in]  Frame normalized world positions
	const float * K_RESTRICT frame_noisy_1spp,				// [in]  Frame noisy 1spp color
		  float * K_RESTRICT frame_acc_noisy,				// [out] Current  frame accumulated noisy color
	const float * K_RESTRICT prev_frame_acc_noisy,			// [in]  Previous frame accumulated noisy color
	const unsigned char * K_RESTRICT prev_frame_acc_spp,	// [in]  Previous frame accumulated number of samples (for CMA)
		  unsigned char * K_RESTRICT frame_acc_num_spp,		// [out] Current  frame accumulated number of samples (for CMA)
	#if USE_HALF_PRECISION_IN_FEATURES_DATA
	half * K_RESTRICT features_data,						// [out] Features buffer (half-precision)
	#else
	float * K_RESTRICT features_data,						// [out] Features buffer (single-precision)
	#endif
	const mat4x4 prev_frame_camera_matrix,					// [in]  ViewProj matrix of previous frame
	const vec2 pixel_offset
)
{
	template_accumulate_noisy_data(
		params,
		out_prev_frame_pixel,
		accept_bools,
		frame_normals,
		prev_frame_normals,
		frame_positions,
		prev_frame_positions,
		frame_normalized_positions,
		frame_noisy_1spp,
		frame_acc_noisy,
		prev_frame_acc_noisy,
		prev_frame_acc_spp,
		frame_acc_num_spp,
		features_data,
		prev_frame_camera_matrix,
		pixel_offset
	);
}

extern "C" void run_new_accumulate_noisy_data(
	dim3 const & grid_size,
	dim3 const & block_size,
	AccumulateNoisyDataKernelParams const & params,
	vec2 * K_RESTRICT out_prev_frame_pixel,					// [out] Previous frame pixel coordinates (after reprojection)
	unsigned char* K_RESTRICT accept_bools,					// [out] Validity mask of bilinear samples in previous frame (after reprojection)
	const float * K_RESTRICT frame_normals,					// [in]  Current  (world) normals
	const float * K_RESTRICT prev_frame_normals,			// [in]  Previous (world) normals
	const float * K_RESTRICT frame_positions,				// [in]  Current  world positions
	const float * K_RESTRICT prev_frame_positions,			// [in]  Previous world positions
	const float * K_RESTRICT frame_normalized_positions,	// [in]  Frame normalized world positions
	const float * K_RESTRICT frame_noisy_1spp,				// [in]  Frame noisy 1spp color buffer
		  float * K_RESTRICT frame_acc_noisy,				// [out] Current  noisy 1spp color
	const float * K_RESTRICT prev_frame_acc_noisy,			// [in]  Previous noisy 1spp color
	const unsigned char * K_RESTRICT prev_frame_acc_spp,	// [in]  Previous number of samples accumulated (for CMA)
		  unsigned char * K_RESTRICT frame_acc_num_spp,		// [out] Current  number of samples accumulated (for CMA)
	#if USE_HALF_PRECISION_IN_FEATURES_DATA
	half * K_RESTRICT features_data,						// [out] Features buffer (half-precision)
	#else
	float * K_RESTRICT features_data,						// [out] Features buffer (single-precision)
	#endif
	const mat4x4 prev_frame_camera_matrix,					// [in]  ViewProj matrix of previous frame
	const vec2 pixel_offset
)
{
	new_accumulate_noisy_data<<<grid_size, block_size>>>(
		params,
		out_prev_frame_pixel,
		accept_bools,
		frame_normals,
		prev_frame_normals,
		frame_positions,
		prev_frame_positions,
		frame_normalized_positions,
		frame_noisy_1spp,
		frame_acc_noisy,
		prev_frame_acc_noisy,
		prev_frame_acc_spp,
		frame_acc_num_spp,
		features_data,
		prev_frame_camera_matrix,
		pixel_offset
	);
}

// Fitter kernel ///////////////////////////////////////////////////////////////

// Fitter kernel ///////////////////////////////////////////////////////////////

#define USE_FEATURES_VGPR_CACHE 1

// Block size: (256, 1, 1)
__global__ void new_fitter(
	FitterKernelParams params,
	float * K_RESTRICT weights,					// [out] Features weights
	#if USE_HALF_PRECISION_IN_FEATURES_DATA
	half * K_RESTRICT features_buffer			// [out] Features buffer (half-precision)
	#else
	float * K_RESTRICT features_buffer			// [out] Features buffer (single-precision)
	#endif
)
{
	// Notes:
	//  LOCAL_SIZE = 256
	//	BLOCK_PIXELS = 32 * 32
	
	// TODO: send as define for cpp side
	#if COMPRESSED_R
    //const auto r_size = ((buffer_count - 2) * (buffer_count - 1) / 2) * sizeof(cl_float3);
	#define R_SHARED_DATA_SIZE ((BUFFER_COUNT - 2) * (BUFFER_COUNT - 1) / 2)
	#else
    //const auto r_size = (buffer_count - 2) * (buffer_count - 2) * sizeof(cl_float3);
	#define R_SHARED_DATA_SIZE ((BUFFER_COUNT - 2) * (BUFFER_COUNT - 2))
	#endif

	__shared__ float pr_shared_data[LOCAL_SIZE];		// Shared memory used to perform parallel reduction
	__shared__ float u_vec_sdata[BLOCK_PIXELS];			// Shared memory used to store the 'u' vectors
	__shared__ cvec3 r_mat_sdata[R_SHARED_DATA_SIZE];	// Shared memory used to store the R matrices of the QR factorization (vec3 -> one per color channel)
	__shared__ float u_length_squared;					// Shared memory variable that holds the 'u' vector square length
	__shared__ float dotProd;							// Shared memory variable that holds the dot product of...
	__shared__ float vec_length;						// Shared memory variable that holds the vec length			

	float * pr_data_256 = &pr_shared_data[0];
	float * u_vec = &u_vec_sdata[0];
	cvec3 * r_mat = &r_mat_sdata[0];

	const int groupId = blockIdx.x;
	const int threadId = threadIdx.x; // in [0, 255]

	const unsigned int blockIndexX = groupId % params.worksetWithMarginBlockCountX;
	const unsigned int blockIndexY = groupId / params.worksetWithMarginBlockCountX;
	const unsigned int linearBlockIndex = blockIndexY * params.worksetWithMarginBlockCountX + blockIndexX;
	const unsigned int threadFeaturesBuffersOffset = linearBlockIndex * BUFFER_COUNT * BLOCK_PIXELS + threadId;

	const unsigned int baseSeed = params.frameNumber * BUFFER_COUNT * BLOCK_PIXELS + threadId;
	
	// Non square matrices require processing every column.
	// Otherwise result is OKish, but R is not upper triangular matrix
	const int limit = (BUFFER_COUNT == BLOCK_PIXELS) ? BUFFER_COUNT - 1 : BUFFER_COUNT;

	
	#if USE_FEATURES_VGPR_CACHE
	const unsigned int FeaturesCacheSize = BLOCK_PIXELS / LOCAL_SIZE * BUFFER_COUNT;
	#if USE_HALF_PRECISION_IN_FEATURES_DATA
	half featuresCache[FeaturesCacheSize];
	#else
	float featuresCache[FeaturesCacheSize];
	#endif

	for(unsigned int featureIndex = 0; featureIndex < BUFFER_COUNT; ++featureIndex)
	{
		const unsigned int baseFeatureOffset = featureIndex * BLOCK_PIXELS + threadFeaturesBuffersOffset;
		const unsigned int baseFeaturesCacheOffset = featureIndex * (BLOCK_PIXELS / LOCAL_SIZE);

		for(int subVector = 0; subVector < BLOCK_PIXELS / LOCAL_SIZE; ++subVector)
		{
			const unsigned int featureOffset = subVector * LOCAL_SIZE + baseFeatureOffset;
			const unsigned int featuresCacheOffset = baseFeaturesCacheOffset + subVector;
			#if USE_HALF_PRECISION_IN_FEATURES_DATA
			featuresCache[featuresCacheOffset] = features_buffer[featureOffset];
			#else
			featuresCache[featuresCacheOffset] = load_feature(features_buffer, featureOffset);
			#endif
		}
	}
	#endif

	// Compute R
	for(int col = 0; col < limit; col++)
	{
		// Note: the last 3 features values are the 3 channels of the color (not used for the regression)
		int col_limited = Min(col, BUFFER_COUNT - 3);

		// Load new column into memory
		const int featureIndex = col;

		#if USE_FEATURES_VGPR_CACHE
		const unsigned int baseFeaturesCacheOffset = featureIndex * (BLOCK_PIXELS / LOCAL_SIZE);
		#else
		const unsigned int baseFeatureOffset = featureIndex * BLOCK_PIXELS + threadFeaturesBuffersOffset;
		#endif

		float tmp_sum_value = 0.f;

		// Manual unrolling for parallel reduction as the block contains 1024 (32x32) work items and
		// the reduction operates on 256 elements (group size)
		// -> Compute the sum of N values (N = 1024/256 = 4)
		for(int subVector = 0; subVector < BLOCK_PIXELS / LOCAL_SIZE; ++subVector)
		{
			// Load feature
			#if USE_FEATURES_VGPR_CACHE
				const unsigned int featuresCacheOffset = baseFeaturesCacheOffset + subVector;
				#if USE_HALF_PRECISION_IN_FEATURES_DATA
				float tmp = HalfToFloat(featuresCache[featuresCacheOffset]);
				#else
				float tmp = featuresCache[featuresCacheOffset];
				#endif
			#else
				const unsigned int featureOffset = subVector * LOCAL_SIZE + baseFeatureOffset;
				float tmp = load_feature(features_buffer, featureOffset);
			#endif

			// Store the feature in shared memory
			const int index = subVector * LOCAL_SIZE + threadId;
			u_vec[index] = tmp;

			if(index >= col_limited + 1)
			{
				tmp_sum_value += tmp * tmp;
			}
		}
		SyncThreads();

		// Find length of vector in A's column with reduction sum function
		pr_data_256[threadId] = tmp_sum_value;
		SyncThreads();
		parallel_reduction_sum_256(&vec_length, pr_data_256, col_limited + 1);

		// NOTE: GCN Opencl compiler can do some optimization with this because if
		// initially wanted col_limited is used to select wich work-item runs which branch
		// it is slower. However using col produces the same result.
		float r_value;
		if(threadId < col)
		{
			// Copy u_vec value
			r_value = u_vec[threadId];
		}
		else if(threadId == col)
		{
			u_length_squared = vec_length;
			vec_length = Sqrt(vec_length + u_vec[col_limited] * u_vec[col_limited]);
			u_vec[col_limited] -= vec_length;
			u_length_squared += u_vec[col_limited] * u_vec[col_limited];

			// (u_length_squared is now updated length squared)
			r_value = vec_length;
		}
		else if(threadId > col) //Could have "&& threadId <  R_EDGE" but this is little bit faster
		{
			// Last values on every column are zeros
			r_value = 0.0f;
		}

		int id_limited = Min(threadId, BUFFER_COUNT - 3);
		if(col < BUFFER_COUNT - 3)
			store_r_mat_broadcast(r_mat, col_limited, id_limited, r_value);
		else
			store_r_mat_channel(r_mat, col_limited, id_limited, col - BUFFER_COUNT + 3, r_value);
		SyncThreads();

		// Transform further columns of A
		// NOTE: three last columns are three color channels of noisy data. However,
		// they all need to be transfomed as they were column indexed (buffers - 3)
		for(int featureIndex = col_limited+1; featureIndex < BUFFER_COUNT; ++featureIndex)
		{
			#if USE_FEATURES_VGPR_CACHE
			const unsigned int baseFeaturesCacheOffset = featureIndex * (BLOCK_PIXELS / LOCAL_SIZE);
			#else
			const unsigned int baseFeatureOffset = featureIndex * BLOCK_PIXELS + threadFeaturesBuffersOffset;
			#endif

			const unsigned int baseFeatureSeed = featureIndex * BLOCK_PIXELS + baseSeed;

			// Starts by computing dot product with reduction sum function
			#if CACHE_TMP_DATA
			// No need to load features_buffer twice because each work-item first copies value for
			// dot product computation and then modifies the same value
			float tmp_data_private_cache[BLOCK_PIXELS / LOCAL_SIZE];
			#endif

			float tmp_sum_value = 0.f;
			for(int subVector = 0; subVector < BLOCK_PIXELS / LOCAL_SIZE; ++subVector)
			{
				const int index = subVector * LOCAL_SIZE + threadId;
				if(index >= col_limited)
				{
					// Load feature

					#if USE_FEATURES_VGPR_CACHE
						const unsigned int featuresCacheOffset = baseFeaturesCacheOffset + subVector;
						#if USE_HALF_PRECISION_IN_FEATURES_DATA
						float tmp = HalfToFloat(featuresCache[featuresCacheOffset]);
						#else
						float tmp = featuresCache[featuresCacheOffset];
						#endif
					#else
						const unsigned int featureOffset = subVector * LOCAL_SIZE + baseFeatureOffset;
						float tmp = load_feature(features_buffer, featureOffset);
					#endif

					// [Section 3.4] - Stochastic regularization
					// To handle rank-deficiency in the T matrix, Add zero-mean noise to the input buffers
					// (the first time values are loaded), which makes them linearly independent.
					// Note: does not Add noise to constant buffer (column 0) and noisy image data (last 3 columns).
					if(col == 0 && featureIndex < BUFFER_COUNT - 3)
					{
						const int seed = subVector * LOCAL_SIZE + baseFeatureSeed;
						tmp += NOISE_AMOUNT * SignedZeroMeanNoise(seed);
					}

					#if CACHE_TMP_DATA
					tmp_data_private_cache[subVector] = tmp;
					#endif
					tmp_sum_value += tmp * u_vec[index];
				}
			}

			pr_data_256[threadId] = tmp_sum_value;
			SyncThreads();
			parallel_reduction_sum_256(&dotProd, pr_data_256, col_limited);

			const float dotFactor = 2.0f * dotProd / u_length_squared;

			// Manual unrolling as the block contains 1024 (32x32) work items and we operate on 256 elements (group size)
			// -> Compute the sum of N values (N = 1024/256 = 4)
			for(int subVector = 0; subVector < BLOCK_PIXELS / LOCAL_SIZE; ++subVector)
			{
				const int index = subVector * LOCAL_SIZE + threadId;
				if(index >= col_limited)
				{
					#if USE_FEATURES_VGPR_CACHE
					const unsigned int featuresCacheOffset = baseFeaturesCacheOffset + subVector;
					#else
					const unsigned int featureOffset = subVector * LOCAL_SIZE + baseFeatureOffset;
					#endif

					#if CACHE_TMP_DATA
					float store_value = tmp_data_private_cache[subVector];
					#else
						#if USE_FEATURES_VGPR_CACHE
							#if USE_HALF_PRECISION_IN_FEATURES_DATA
							float store_value = HalfToFloat(featuresCache[featuresCacheOffset]);
							#else
							float store_value = featuresCache[featuresCacheOffset];
							#endif
						#else
							float store_value = load_feature(features_buffer, featureOffset);
						#endif
					const int seed = subVector * LOCAL_SIZE + baseFeatureSeed;
					store_value += NOISE_AMOUNT * SignedZeroMeanNoise(seed);
					#endif 

					store_value -= dotFactor * u_vec[index];

					#if USE_FEATURES_VGPR_CACHE
						#if USE_HALF_PRECISION_IN_FEATURES_DATA
						featuresCache[featuresCacheOffset] = FloatToHalf(store_value);
						#else
						featuresCache[featuresCacheOffset] = store_value;
						#endif
					#else
						store_feature(features_buffer, featureOffset, store_value);
					#endif
				}
			}
			#if !USE_FEATURES_VGPR_CACHE
			GlobalMemFence();
			#endif
		}
	}

	// Back substitution
	__shared__ cvec3 divider; // Shared memory variable that holds the divider

	// R_EDGE = buffer_count - 2 (= number of features + 3 (noisy color spp buffer) - 2)
	// R is a (M + 1)x(M + 1) matrix, with M the number of features (here equal to buffer_count - 3)
	// which gives us R_EDGE = M + 1 = buffer_count - 3 + 1 = buffer_count - 2
	for(int i = R_EDGE - 2; i >= 0; i--)
	{
		if(threadId == 0)
			divider = load_r_mat(r_mat, i, i);
		
		SyncThreads();
		
		#if COMPRESSED_R
		if(threadId < R_EDGE && threadId >= i)
		#else
		// First values are always zero if R !COMPRESSED_R and
		// "&& threadId >= i" makes not compressed code run little bit slower
		if(threadId < R_EDGE)
		#endif
		{
			vec3 value = load_r_mat(r_mat, threadId, i);
			store_r_mat(r_mat, threadId, i, value / vec3(divider.x, divider.y, divider.z));
		}

		SyncThreads();

		#if 1 // ORIGINAL
		if(threadId == 0) // Optimization proposal: parallel reduction
		{
			for(int j = i + 1; j < R_EDGE - 1; j++)
			{
				vec3 value  = load_r_mat(r_mat, R_EDGE - 1, i);
				vec3 value2 = load_r_mat(r_mat, j, i);
				store_r_mat(r_mat, R_EDGE - 1, i, value - value2);
			}
		}
		#else
		const int startRIdx = (i + 1);
		const int endRIdx	= (R_EDGE - 1);
		const int numItems	= endRIdx - startRIdx;
		if(threadId < numItems)
		{
			// Parallel load
			const int j = startRIdx + threadId;
			vec3 value2 = load_r_mat(r_mat, j, i);

			// Then iterate over active threads and gather data from lane 0
			if(threadId == 0)
			{
				for(int k = startRIdx; k < endRIdx; k++)
				{
					vec3 value = load_r_mat(r_mat, R_EDGE - 1, i);
					vec3 currValue2 = ...;// load from lane k
					store_r_mat(r_mat, R_EDGE - 1, i, value - currValue2);
				}
			}
		}
		#endif

		SyncThreads();

		#if COMPRESSED_R
		if(threadId < R_EDGE && i >= threadId)
		#else
		if(threadId < R_EDGE)
		#endif
		{
			vec3 value  = load_r_mat(r_mat, i, threadId);
			vec3 value2 = load_r_mat(r_mat, R_EDGE - 1, i);
			store_r_mat(r_mat, i, threadId, value * value2);
		}
		SyncThreads();
	}

	// The features are stored in the first (buffers-3) values: the last 3 contain the noisy 1spp color channels
	if(threadId < BUFFER_COUNT - 3)
	{
		// Store weights
		const int index = groupId * (BUFFER_COUNT - 3) + threadId;
		const vec3 weight = load_r_mat(r_mat, R_EDGE - 1, threadId);
		store_float3(weights, index, weight);
	}
}

extern "C" void run_new_fitter(
	dim3 const & grid_size,
	dim3 const & block_size,
	FitterKernelParams const & params,
	float * K_RESTRICT weights,					// [out] Features weights
	#if USE_HALF_PRECISION_IN_FEATURES_DATA
	half * K_RESTRICT features_buffer			// [out] Features buffer (half-precision)
	#else
	float * K_RESTRICT features_buffer			// [out] Features buffer (single-precision)
	#endif
)
{
	new_fitter<<<grid_size, block_size>>>(
		params,
		weights,
		features_buffer
	);
}

// Unrolled parallel sum reduction of 256 values (half-precision)
// TODO: unused start_index...
inline __device__ void parallel_reduction_sum_256(half * K_RESTRICT result, half * K_RESTRICT pr_data_256, const int start_index)
{
	const int id = threadIdx.x;

	#if K_SUPPORT_HALF16_ARITHMETIC
	if(id < 64)
	{
		half2 tmp = __hadd2(__halves2half2(pr_data_256[id],		  pr_data_256[id + 64]),
							__halves2half2(pr_data_256[id + 128], pr_data_256[id + 192])
					);

		pr_data_256[id] = __hadd(__high2half(tmp), __low2half(tmp));
	}
							
	SyncThreads();

	if(id < 8)
	{
		half2 tmp0 = __hadd2(__halves2half2(pr_data_256[id],	  pr_data_256[id + 8]),
							 __halves2half2(pr_data_256[id + 16], pr_data_256[id + 24])
					 );

		half2 tmp1 = __hadd2(__halves2half2(pr_data_256[id + 32], pr_data_256[id + 40]),
							 __halves2half2(pr_data_256[id + 48], pr_data_256[id + 56])
					 );

		half2 tmp2 = __hadd2(tmp0, tmp1);

		pr_data_256[id] = __hadd(__high2half(tmp2), __low2half(tmp2));
	}
	SyncThreads();

	if(id == 0)
	{
		#if 0
		half2 tmp0 = __hadd2(__halves2half2(pr_data_256[0], pr_data_256[1]),
							 __halves2half2(pr_data_256[2], pr_data_256[3])
					 );

		half2 tmp1 = __hadd2(__halves2half2(pr_data_256[4], pr_data_256[5]),
							 __halves2half2(pr_data_256[6], pr_data_256[7])
					 );
		#else
		half2 tmp0 = __hadd2(*reinterpret_cast<half2*>(pr_data_256 + 0),
							 *reinterpret_cast<half2*>(pr_data_256 + 2)
					 );
		half2 tmp1 = __hadd2(*reinterpret_cast<half2*>(pr_data_256 + 4),
							 *reinterpret_cast<half2*>(pr_data_256 + 6)
					 );
		#endif

		half2 tmp2 = __hadd2(tmp0, tmp1);

		*result = __hadd(__high2half(tmp2), __low2half(tmp2));
	}
	SyncThreads();
	#else // K_SUPPORT_HALF16_ARITHMETIC
	if(id < 64)
	{
		pr_data_256[id] = Add(
							Add(pr_data_256[id],		pr_data_256[id + 64]),
							Add(pr_data_256[id + 128],	pr_data_256[id + 192])
						  );
	}
	SyncThreads();

	if(id < 8)
	{
		pr_data_256[id] = Add(
							Add(Add(pr_data_256[id], pr_data_256[id + 8]),		 Add(pr_data_256[id + 16], pr_data_256[id + 24])),
							Add(Add(pr_data_256[id + 32], pr_data_256[id + 40]), Add(pr_data_256[id + 48], pr_data_256[id + 56]))
						  );
	}
	SyncThreads();

	if(id == 0)
	{
		*result = Add(
					Add(Add(pr_data_256[0], pr_data_256[1]), Add(pr_data_256[2], pr_data_256[3])),
					Add(Add(pr_data_256[4], pr_data_256[5]), Add(pr_data_256[6], pr_data_256[7]))
				  );
	}
	SyncThreads();
#endif // K_SUPPORT_HALF16_ARITHMETIC
}


inline __device__ void load_r_mat(const half * K_RESTRICT r_mat, unsigned int x, unsigned int y, half oValue[3])
{
	const unsigned int offset = 3 * R_ACCESS;
	oValue[0] = r_mat[offset + 0];
	oValue[1] = r_mat[offset + 1];
	oValue[2] = r_mat[offset + 2];
}

inline __device__ void store_r_mat(half * K_RESTRICT r_mat, unsigned int x, unsigned int y, half value[3])
{
	const unsigned int offset = 3 * R_ACCESS;
	r_mat[offset + 0] = value[0];
	r_mat[offset + 1] = value[1];
	r_mat[offset + 2] = value[2];
}

inline __device__ void store_r_mat_broadcast(half * r_mat, unsigned int x, unsigned int y, half value)
{
	const unsigned int offset = 3 * R_ACCESS;
	r_mat[offset + 0] = value;
	r_mat[offset + 1] = value;
	r_mat[offset + 2] = value;
}

inline __device__ void store_r_mat_channel(half * r_mat, unsigned int x, unsigned int y, unsigned int channel, half value)
{
	const unsigned int offset = 3 * R_ACCESS + channel;
	r_mat[offset] = value;
}

// Block size: (256, 1, 1)
__global__ void fitter16bits(
	FitterKernelParams params,
	//half * K_RESTRICT weights,			// [out] Features weights
	float * K_RESTRICT weights,			// [out] Features weights
	half * K_RESTRICT features_buffer	// [in]  Features buffer
)
{
	// Notes:
	//  LOCAL_SIZE = 256
	//	BLOCK_PIXELS = 32 * 32
	
	// TODO: send as define for cpp side
	#if COMPRESSED_R
    //const auto r_size = ((buffer_count - 2) * (buffer_count - 1) / 2) * sizeof(cl_float3);
	#define R_SHARED_DATA_SIZE ((BUFFER_COUNT - 2) * (BUFFER_COUNT - 1) / 2)
	#else
    //const auto r_size = (buffer_count - 2) * (buffer_count - 2) * sizeof(cl_float3);
	#define R_SHARED_DATA_SIZE ((BUFFER_COUNT - 2) * (BUFFER_COUNT - 2))
	#endif

	__shared__ half  pr_shared_data[LOCAL_SIZE];			// Shared memory used to perform parallel reduction
	__shared__ half  u_vec_sdata[BLOCK_PIXELS];				// Shared memory used to store the 'u' vectors
	__shared__ half  r_mat_sdata[3 * R_SHARED_DATA_SIZE];	// Shared memory used to store the R matrices of the QR factorization (x3 -> one per color channel)
	__shared__ half u_length_squared;						// Shared memory variable that holds the 'u' vector square length
	__shared__ half dotProd;								// Shared memory variable that holds the dot product of...
	__shared__ half vec_length;								// Shared memory variable that holds the vec length			

	half * pr_data_256 = &pr_shared_data[0];
	half * u_vec = &u_vec_sdata[0];
	half * r_mat = &r_mat_sdata[0];

	const int groupId = blockIdx.x;
	const int threadId = threadIdx.x; // in [0, 255]

	const unsigned int blockIndexX = groupId % params.worksetWithMarginBlockCountX;
	const unsigned int blockIndexY = groupId / params.worksetWithMarginBlockCountX;
	const unsigned int linearBlockIndex = blockIndexY * params.worksetWithMarginBlockCountX + blockIndexX;
	const unsigned int threadFeaturesBuffersOffset = linearBlockIndex * BUFFER_COUNT * BLOCK_PIXELS + threadId;

	const unsigned int baseSeed = params.frameNumber * BUFFER_COUNT * BLOCK_PIXELS + threadId;
	
	// Non square matrices require processing every column.
	// Otherwise result is OKish, but R is not upper triangular matrix
	const int limit = (BUFFER_COUNT == BLOCK_PIXELS) ? BUFFER_COUNT - 1 : BUFFER_COUNT;

	
	#if USE_FEATURES_VGPR_CACHE
	const unsigned int FeaturesCacheSize = BLOCK_PIXELS / LOCAL_SIZE * BUFFER_COUNT;
	half featuresCache[FeaturesCacheSize];

	for(unsigned int featureIndex = 0; featureIndex < BUFFER_COUNT; ++featureIndex)
	{
		const unsigned int baseFeatureOffset = featureIndex * BLOCK_PIXELS + threadFeaturesBuffersOffset;
		const unsigned int baseFeaturesCacheOffset = featureIndex * (BLOCK_PIXELS / LOCAL_SIZE);

		for(int subVector = 0; subVector < BLOCK_PIXELS / LOCAL_SIZE; ++subVector)
		{
			const unsigned int featureOffset = subVector * LOCAL_SIZE + baseFeatureOffset;
			const unsigned int featuresCacheOffset = baseFeaturesCacheOffset + subVector;
			featuresCache[featuresCacheOffset] = features_buffer[featureOffset];
		}
	}
	#endif

	// Compute R
	for(int col = 0; col < limit; col++)
	{
		// Note: the last 3 features values are the 3 channels of the color (not used for the regression)
		int col_limited = Min(col, BUFFER_COUNT - 3);

		// Load new column into memory
		const int featureIndex = col;
		
		#if USE_FEATURES_VGPR_CACHE
		const unsigned int baseFeaturesCacheOffset = featureIndex * (BLOCK_PIXELS / LOCAL_SIZE);
		#else
		const unsigned int baseFeatureOffset = featureIndex * BLOCK_PIXELS + threadFeaturesBuffersOffset;
		#endif

		half tmp_sum_value = FloatToHalf(0.0f);

		// Manual unrolling for parallel reduction as the block contains 1024 (32x32) work items and
		// the reduction operates on 256 elements (group size)
		// -> Compute the sum of N values (N = 1024/256 = 4)
		for(int subVector = 0; subVector < BLOCK_PIXELS / LOCAL_SIZE; ++subVector)
		{
			// Load feature
			#if USE_FEATURES_VGPR_CACHE
			const unsigned int featuresCacheOffset = baseFeaturesCacheOffset + subVector;
			half tmp = featuresCache[featuresCacheOffset];
			#else
			const unsigned int featureOffset = subVector * LOCAL_SIZE + baseFeatureOffset;
			half tmp = features_buffer[featureOffset];
			#endif

			// Store the feature in shared memory
			const int index = subVector * LOCAL_SIZE + threadId;
			u_vec[index] = tmp;

			if(index >= col_limited + 1)
			{
				tmp_sum_value = Add(tmp_sum_value, Mul(tmp, tmp));
			}
		}
		SyncThreads();

		// Find length of vector in A's column with reduction sum function
		pr_data_256[threadId] = tmp_sum_value;
		SyncThreads();
		parallel_reduction_sum_256(&vec_length, pr_data_256, col_limited + 1);

		// NOTE: GCN Opencl compiler can do some optimization with this because if
		// initially wanted col_limited is used to select wich work-item runs which branch
		// it is slower. However using col produces the same result.
		half r_value;
		if(threadId < col)
		{
			// Copy u_vec value
			r_value = u_vec[threadId];
		}
		else if(threadId == col)
		{
			u_length_squared = vec_length;
			vec_length = Sqrt(Add(vec_length, Mul(u_vec[col_limited], u_vec[col_limited])));
			u_vec[col_limited] = Sub(u_vec[col_limited], vec_length);
			u_length_squared = Add(u_length_squared, Mul(u_vec[col_limited], u_vec[col_limited]));

			// (u_length_squared is now updated length squared)
			r_value = vec_length;
		}
		else if(threadId > col) //Could have "&& threadId <  R_EDGE" but this is little bit faster
		{
			// Last values on every column are zeros
			r_value = FloatToHalf(0.0f);
		}

		int id_limited = Min(threadId, BUFFER_COUNT - 3);
		if(col < BUFFER_COUNT - 3)
			store_r_mat_broadcast(r_mat, col_limited, id_limited, r_value);
		else
			store_r_mat_channel(r_mat, col_limited, id_limited, col - BUFFER_COUNT + 3, r_value);
		SyncThreads();

		// Transform further columns of A
		// NOTE: three last columns are three color channels of noisy data. However,
		// they all need to be transfomed as they were column indexed (buffers - 3)
		for(int featureIndex = col_limited+1; featureIndex < BUFFER_COUNT; ++featureIndex)
		{
			#if USE_FEATURES_VGPR_CACHE
			const unsigned int baseFeaturesCacheOffset = featureIndex * (BLOCK_PIXELS / LOCAL_SIZE);
			#else
			const unsigned int baseFeatureOffset = featureIndex * BLOCK_PIXELS + threadFeaturesBuffersOffset;
			#endif

			const unsigned int baseFeatureSeed = featureIndex * BLOCK_PIXELS + baseSeed;

			// Starts by computing dot product with reduction sum function
			#if CACHE_TMP_DATA
			// No need to load features_buffer twice because each work-item first copies value for
			// dot product computation and then modifies the same value
			half tmp_data_private_cache[BLOCK_PIXELS / LOCAL_SIZE];
			#endif

			half tmp_sum_value = FloatToHalf(0.0f);
			for(int subVector = 0; subVector < BLOCK_PIXELS / LOCAL_SIZE; ++subVector)
			{
				const int index = subVector * LOCAL_SIZE + threadId;
				if(index >= col_limited)
				{
					// Load feature
					#if USE_FEATURES_VGPR_CACHE
					const unsigned int featuresCacheOffset = baseFeaturesCacheOffset + subVector;
					half tmp = featuresCache[featuresCacheOffset];
					#else
					const unsigned int featureOffset = subVector * LOCAL_SIZE + baseFeatureOffset;
					half tmp = features_buffer[featureOffset];
					#endif

					// [Section 3.4] - Stochastic regularization
					// To handle rank-deficiency in the T matrix, Add zero-mean noise to the input buffers
					// (the first time values are loaded), which makes them linearly independent.
					// Note: does not Add noise to constant buffer (column 0) and noisy image data (last 3 columns).
					if(col == 0 && featureIndex < BUFFER_COUNT - 3)
					{
						const int seed = subVector * LOCAL_SIZE + baseFeatureSeed;
						tmp = Add(tmp, FloatToHalf(NOISE_AMOUNT * SignedZeroMeanNoise(seed)));
					}

					#if CACHE_TMP_DATA
					tmp_data_private_cache[subVector] = tmp;
					#endif
					tmp_sum_value = Add(tmp_sum_value, Mul(tmp, u_vec[index]));
				}
			}

			pr_data_256[threadId] = tmp_sum_value;
			SyncThreads();
			parallel_reduction_sum_256(&dotProd, pr_data_256, col_limited);

			const half dotFactor = Mul(FloatToHalf(2.0f), Div(dotProd, u_length_squared));

			// Manual unrolling as the block contains 1024 (32x32) work items and we operate on 256 elements (group size)
			// -> Compute the sum of N values (N = 1024/256 = 4)
			for(int subVector = 0; subVector < BLOCK_PIXELS / LOCAL_SIZE; ++subVector)
			{
				const int index = subVector * LOCAL_SIZE + threadId;
				if(index >= col_limited)
				{
					#if USE_FEATURES_VGPR_CACHE
					const unsigned int featuresCacheOffset = baseFeaturesCacheOffset + subVector;
					#else
					const unsigned int featureOffset = subVector * LOCAL_SIZE + baseFeatureOffset;
					#endif

					#if CACHE_TMP_DATA
						half store_value = tmp_data_private_cache[subVector];
					#else
						#if USE_FEATURES_VGPR_CACHE
						half store_value = featuresCache[featuresCacheOffset];
						#else
						half store_value = features_buffer[featureOffset];
						#endif
						const int seed = subVector * LOCAL_SIZE + baseFeatureSeed;
						store_value = Add(store_value, FloatToHalf(NOISE_AMOUNT * SignedZeroMeanNoise(seed)));
					#endif 

					store_value = Sub(store_value, Mul(dotFactor, u_vec[index]));

					#if USE_FEATURES_VGPR_CACHE
					featuresCache[featuresCacheOffset] = store_value;
					#else
					features_buffer[featureOffset] = store_value;
					#endif
				}
			}
			#if !USE_FEATURES_VGPR_CACHE
			GlobalMemFence();
			#endif
		}
	}

	// Back substitution
	__shared__ half divider[3]; // Shared memory variable that holds the divider

	// R_EDGE = buffer_count - 2 (= number of features + 3 (noisy color spp buffer) - 2)
	// R is a (M + 1)x(M + 1) matrix, with M the number of features (here equal to buffer_count - 3)
	// which gives us R_EDGE = M + 1 = buffer_count - 3 + 1 = buffer_count - 2
	for(int i = R_EDGE - 2; i >= 0; i--)
	{
		if(threadId == 0)
			load_r_mat(r_mat, i, i, divider);
		
		SyncThreads();
		
		#if COMPRESSED_R
		if(threadId < R_EDGE && threadId >= i)
		#else
		// First values are always zero if R !COMPRESSED_R and
		// "&& threadId >= i" makes not compressed code run little bit slower
		if(threadId < R_EDGE)
		#endif
		{
			half value[3];
			load_r_mat(r_mat, threadId, i, value);
			Div(value, divider, value);
			store_r_mat(r_mat, threadId, i, value);
		}

		SyncThreads();

		if(threadId == 0) // Optimization proposal: parallel reduction
		{
			for(int j = i + 1; j < R_EDGE - 1; j++)
			{
				half value[3];
				load_r_mat(r_mat, R_EDGE - 1, i, value);
				half value2[3];
				load_r_mat(r_mat, j, i, value2);
				Sub(value, value2, value);
				store_r_mat(r_mat, R_EDGE - 1, i, value);
			}
		}

		SyncThreads();

		#if COMPRESSED_R
		if(threadId < R_EDGE && i >= threadId)
		#else
		if(threadId < R_EDGE)
		#endif
		{
			half value[3];
			load_r_mat(r_mat, i, threadId, value);
			half value2[3];
			load_r_mat(r_mat, R_EDGE - 1, i, value2);
			Mul(value, value2, value);
			store_r_mat(r_mat, i, threadId, value);
		}
		SyncThreads();
	}

	// The features are stored in the first (buffers-3) values: the last 3 contain the noisy 1spp color channels
	if(threadId < BUFFER_COUNT - 3)
	{
		// Store weights
		const int index = groupId * (BUFFER_COUNT - 3) + threadId;
		half weight[3];
		load_r_mat(r_mat, R_EDGE - 1, threadId, weight);
		vec3 fweight;
		fweight.x = HalfToFloat(weight[0]);
		fweight.y = HalfToFloat(weight[1]);
		fweight.z = HalfToFloat(weight[2]);
		store3(weights, index, fweight);
	}
}


extern "C" void run_fitter16bits(
	dim3 const & grid_size,
	dim3 const & block_size,
	FitterKernelParams params,
	//half * K_RESTRICT weights,			// [out] Features weights
	float * K_RESTRICT weights,			// [out] Features weights
	half * K_RESTRICT features_buffer	// [in]  Features buffer
)
{
	fitter16bits<<<grid_size, block_size>>>(params, weights, features_buffer);
}

// Weighted sum kernel /////////////////////////////////////////////////////////
// -> outputs the noise-free 1spp color estimate

__global__ void new_weighted_sum(
	WeightedSumKernelParams params,
	const float * K_RESTRICT weights,			// [in]	 Features weights computed by the fitter kernel
		  float * K_RESTRICT output,			// [out] Noise-free color estimate
	const float * K_RESTRICT current_normals,	// [in]  Current (world) normals
	const float * K_RESTRICT normalized_world_positions	// [in]  Current world positions
)
{
	const ivec2 pixel = ivec2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	
	const int w = params.sizeX;
	const int h = params.sizeY;

	if(pixel.x >= w || pixel.y >= h)
		return;

	// Linear pixel index
	const int linear_pixel = pixel.y * w + pixel.x;

	// Retrieve linear group index from the offset pixel
	const ivec2 offset_pixel = pixel + BLOCK_EDGE_HALF - BLOCK_OFFSETS[params.frameNumber % BLOCK_OFFSETS_COUNT];
	const int group_index = (offset_pixel.x / BLOCK_EDGE_LENGTH) + (offset_pixel.y / BLOCK_EDGE_LENGTH) * params.worksetWithMarginBlockCountX;

	// Reload features from buffer here to have values without stochastic regularization noise
	// TODO: bind the normalized world_position buffer to avoid renormalizing again (no need for mins_maxs buffer)
	vec3 normalized_world_position = load3<float>(normalized_world_positions, linear_pixel); 
	vec3 normal = load3<float>(current_normals, linear_pixel);

	float features[BUFFER_COUNT-3];
	compute_features_without_color(normalized_world_position, normal, features);

	const unsigned baseWeightOffset = group_index * (BUFFER_COUNT - 3);

	// Weighted sum of the feature buffers
	vec3 color = vec3(0.f, 0.f, 0.f);
	for(int feature_buffer = 0; feature_buffer < BUFFER_COUNT - 3; feature_buffer++)
	{
		float feature = features[feature_buffer];
		vec3 weight = load3<float>(weights, baseWeightOffset + feature_buffer);
		color += weight * feature;
	}

	// Remove negative values from every component of the fitting results
	color = Max(vec3(0.f), color); // TODO -Min(-color, vec3(0.f));

	// Store results
	store3(output, linear_pixel, color);
}

extern "C" void run_new_weighted_sum(
	dim3 const & grid_size,
	dim3 const & block_size,
	WeightedSumKernelParams const & params,
	const float * K_RESTRICT weights,			// [in]	 Features weights computed by the fitter kernel
		  float * K_RESTRICT output,			// [out] Noise-free color estimate
	const float * K_RESTRICT current_normals,	// [in]  Current (world) normals
	const float * K_RESTRICT current_positions	// [in]  Current world positions
)
{
	new_weighted_sum<<<grid_size, block_size>>>(
		params,
		weights,
		output,
		current_normals,
		current_positions
	);
}

// Accumulate filtered data kernel /////////////////////////////////////////////
// -> outputs the noise-free accumulated color estimate + a tonemapped version w/ albedo

__global__ void accumulate_filtered_data_frame0(
	AccumulateFilteredDataKernelParams2 params,
	const float * K_RESTRICT filtered_frame,			// [in]  Noise free color estimate (computed as the weighted sum of the features)
	const float * K_RESTRICT albedo_buffer,				// [in]  Albedo buffer of the current frame (non-noisy)
		  float * K_RESTRICT tone_mapped_frame,			// [out] Accumulated and tonemapped noise-free color estimate
		  float * K_RESTRICT accumulated_frame			// [out] Current frame noise-free accumulated color estimate
)
{
	const ivec2 pixel = ivec2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	
	const int w = params.sizeX;
	const int h = params.sizeY;
	
	if(pixel.x >= w || pixel.y >= h)
		return;

	// Linear pixel index
	const unsigned int linear_pixel = pixel.y * w + pixel.x;

	// Noise-free estimate of the color (computed via a weighted sum of features)
	vec3 filtered_color = load3<float>(filtered_frame, linear_pixel);
	store3(accumulated_frame, linear_pixel, filtered_color);

	// Remodulate albedo and tone map
	vec3 albedo = load3<float>(albedo_buffer, linear_pixel);
	const vec3 tone_mapped_color = Clamp(Pow(Max(vec3(0.f), albedo * filtered_color), 0.454545f), vec3(0.f), vec3(1.f));
	store3(tone_mapped_frame, linear_pixel, tone_mapped_color);
}

extern "C" void run_accumulate_filtered_data_frame0(
	dim3 const & grid_size,
	dim3 const & block_size,
	AccumulateFilteredDataKernelParams2 const & params,
	const float * K_RESTRICT filtered_frame,			// [in]  Noise free color estimate (computed as the weighted sum of the features)
	const float * K_RESTRICT albedo_buffer,				// [in]  Albedo buffer of the current frame (non-noisy)
		  float * K_RESTRICT tone_mapped_frame,			// [out] Accumulated and tonemapped noise-free color estimate
		  float * K_RESTRICT accumulated_frame			// [out] Current frame noise-free accumulated color estimate
)
{
	accumulate_filtered_data_frame0<<<grid_size, block_size>>>(
		params,
		filtered_frame,
		albedo_buffer,
		tone_mapped_frame,
		accumulated_frame
	);
}


__global__ void new_accumulate_filtered_data(
	AccumulateFilteredDataKernelParams2 params,
	const float * K_RESTRICT filtered_frame,			// [in]  Noise free color estimate (computed as the weighted sum of the features)
	const vec2 * K_RESTRICT in_prev_frame_pixel,		// [in]  Previous frame pixel coordinates (after reprojection)
	const unsigned char * K_RESTRICT accept_bools,		// [in]  Validity mask of bilinear samples in previous frame (after reprojection)
	const float * K_RESTRICT albedo_buffer,				// [in]  Albedo buffer of the current frame (non-noisy)
		  float * K_RESTRICT tone_mapped_frame,			// [out] Accumulated and tonemapped noise-free color estimate
	const unsigned char* K_RESTRICT current_spp,		// [in]	 Current number of samples accumulated (for CMA)
	const float * K_RESTRICT accumulated_prev_frame,	// [in]  Previous frame noise-free accumulated color estimate 
		  float * K_RESTRICT accumulated_frame			// [out] Current frame noise-free accumulated color estimate
)
{
	const ivec2 pixel = ivec2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	
	const int w = params.sizeX;
	const int h = params.sizeY;
	
	if(pixel.x >= w || pixel.y >= h)
		return;

	// Linear pixel index
	const unsigned int linear_pixel = pixel.y * w + pixel.x;

	// Noise-free estimate of the color (computed via a weighted sum of features)
	vec3 filtered_color = load3<float>(filtered_frame, linear_pixel);
	vec3 prev_color = vec3(0.f, 0.f, 0.f);
	float blend_alpha = 1.f;

	// Reproject and accumulate previous frame noise-free estimate

	// Bitmask telling which bilinear samples were accepted in the first accumulation kernel
	const unsigned char accept = accept_bools[linear_pixel];

	if(accept > 0) // If any prev frame sample is accepted
	{
		// Pixel coordinates in the previous frame (in [0, w-1]x[0, h-1])
		const vec2 prev_frame_pixel_f = in_prev_frame_pixel[linear_pixel];
			
		// Integer pixel coordinates in the previous frame
		const ivec2 prev_frame_pixel_i = FloatToIntRd(prev_frame_pixel_f);

		// Compute bilinear weights for bilinear sampling
		const vec2 prev_pixel_fract = prev_frame_pixel_f - vec2(prev_frame_pixel_i);
		const vec2 one_minus_prev_pixel_fract = 1.f - prev_pixel_fract;

		float total_weight = 0.f;

		// Add valid bilinear samples

		if(accept & 0x01)
		{
			float weight = one_minus_prev_pixel_fract.x * one_minus_prev_pixel_fract.y;
			int linear_sample_location = prev_frame_pixel_i.y * w + prev_frame_pixel_i.x;
			prev_color += weight * load3<float>(accumulated_prev_frame, linear_sample_location);
			total_weight += weight;
		}

		if(accept & 0x02)
		{
			float weight = prev_pixel_fract.x * one_minus_prev_pixel_fract.y;
			int linear_sample_location = prev_frame_pixel_i.y * w + prev_frame_pixel_i.x + 1;
			prev_color += weight * load3<float>(accumulated_prev_frame, linear_sample_location);
			total_weight += weight;
		}

		if(accept & 0x04)
		{
			float weight = one_minus_prev_pixel_fract.x * prev_pixel_fract.y;
			int linear_sample_location = (prev_frame_pixel_i.y + 1) * w + prev_frame_pixel_i.x;
			prev_color += weight * load3<float>(accumulated_prev_frame, linear_sample_location);
			total_weight += weight;
		}

		if(accept & 0x08)
		{
			float weight = prev_pixel_fract.x * prev_pixel_fract.y;
			int linear_sample_location = (prev_frame_pixel_i.y + 1) * w + prev_frame_pixel_i.x + 1;
			prev_color += weight * load3<float>(accumulated_prev_frame, linear_sample_location);
			total_weight += weight;
		}

		if(total_weight > 0.f)
		{
			// Blend_alpha is dymically decided so that the result is average
			// of all samples (cumulative moving average) until the cap defined by
			// SECOND_BLEND_ALPHA is reached (exponential moving average: EMA_(n+1) = (1 - a) * EMA_n + a * x_(n+1) = lerp(EMA_n, x_(n+1), a))

			// [Section 3.5]
			// Similarly to the first temporal accumulation we use the cumulative moving average until
			// the weight of the new sample has reached the chosen SECOND_BLEND_ALPHA (e.g 10%).
			// Using the cumulative moving average in this second temporal accumulation is crucial since
			// the first block fitted after an occlusion is more likely to contain outlier data and with
			// the cumulative moving average it is mixed with subsequent frames more quickly.
			blend_alpha = 1.f / float(current_spp[linear_pixel]);
			blend_alpha = Max(blend_alpha, SECOND_BLEND_ALPHA);
			prev_color /= total_weight;

			// Note: we accumulate at most 255 samples for the cumulative moving average (which is more than enough because of
			// the threshold SECOND_BLEND_ALPHA that switch to exponential moving average).
			// E.g: SECOND_BLEND_ALPHA = 0.1 = 1 / (n + 1) <=> n = (1 - 0.1) / 0.1 = 9 => above 9 samples for a pixel,
			// we switch to exponential moving average with alpha = 10%
		}
	}

	// Mix with colors and store results
	vec3 accumulated_color = blend_alpha * filtered_color + (1.f - blend_alpha) * prev_color; // Lerp(prev_color, filtered_color, blend_alpha);
	store3(accumulated_frame, linear_pixel, accumulated_color);

	// Remodulate albedo and tone map
	vec3 albedo = load3<float>(albedo_buffer, linear_pixel);
	const vec3 tone_mapped_color = Clamp(Pow(Max(vec3(0.f), albedo * accumulated_color), 0.454545f), vec3(0.f), vec3(1.f));
	store3(tone_mapped_frame, linear_pixel, tone_mapped_color);
}

extern "C" void run_new_accumulate_filtered_data(
	dim3 const & grid_size,
	dim3 const & block_size,
	AccumulateFilteredDataKernelParams2 const & params,
	const float * K_RESTRICT filtered_frame,			// [in]  Noise free color estimate (computed as the weighted sum of the features)
	const vec2 * K_RESTRICT in_prev_frame_pixel,		// [in]  Previous frame pixel coordinates (after reprojection)
	const unsigned char * K_RESTRICT accept_bools,		// [in]  Validity mask of bilinear samples in previous frame (after reprojection)
	const float * K_RESTRICT albedo_buffer,				// [in]  Albedo buffer of the current frame (non-noisy)
		  float * K_RESTRICT tone_mapped_frame,			// [out] Accumulated and tonemapped noise-free color estimate
	const unsigned char* K_RESTRICT current_spp,		// [in]	 Current number of samples accumulated (for CMA)
	const float * K_RESTRICT accumulated_prev_frame,	// [in]  Previous frame noise-free accumulated color estimate 
		  float * K_RESTRICT accumulated_frame			// [out] Current frame noise-free accumulated color estimate
)
{
	new_accumulate_filtered_data<<<grid_size, block_size>>>(
		params,
		filtered_frame,
		in_prev_frame_pixel,
		accept_bools,
		albedo_buffer,
		tone_mapped_frame,
		current_spp,
		accumulated_prev_frame,
		accumulated_frame
	);
}


// TAA kernel //////////////////////////////////////////////////////////////////

// TODO:
// - optimize with local/shared memory
__global__ void taa_frame0(
	TAAKernelParams params,
	const float * K_RESTRICT new_frame,				// [in]	 Current frame color buffer
		  float * K_RESTRICT result_frame			// [out] Antialiased frame color buffer
)
{
	const ivec2 pixel = ivec2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
   
	const int w = params.sizeX;
	const int h = params.sizeY;

	if(pixel.x >= w || pixel.y >= h)
		return;

	// Linear pixel index
	const unsigned int linear_pixel = pixel.y * w + pixel.x;

	store3(result_frame, linear_pixel, load3<float>(new_frame, linear_pixel));
}

extern "C" void run_taa_frame0(
	dim3 const & grid_size,
	dim3 const & block_size,
	TAAKernelParams const & params,
	const float * K_RESTRICT new_frame,				// [in]	 Current frame color buffer
		  float * K_RESTRICT result_frame			// [out] Antialiased frame color buffer
)
{
	taa_frame0<<<grid_size, block_size>>>(params, new_frame, result_frame);
}


__global__ void new_taa(
	TAAKernelParams params,
	const vec2 * K_RESTRICT in_prev_frame_pixel,	// [in]  Previous frame pixel coordinates (after reprojection)
	const float * K_RESTRICT new_frame,				// [in]	 Current frame color buffer
		  float * K_RESTRICT result_frame,			// [out] Antialiased frame color buffer
	const float * K_RESTRICT prev_frame				// [in]  Previous frame color buffer
)
{
	const ivec2 pixel = ivec2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
   
	const int w = params.sizeX;
	const int h = params.sizeY;

	if(pixel.x >= w || pixel.y >= h)
		return;

	// Linear pixel index
	const unsigned int linear_pixel = pixel.y * w + pixel.x;

	// Current frame color
	vec3 my_new_color =	load3<float>(new_frame, linear_pixel);

	// Previous frame pixel coordinates
	const vec2 prev_frame_pixel_f = in_prev_frame_pixel[linear_pixel];
	const ivec2 prev_frame_pixel_i = FloatToIntRd(prev_frame_pixel_f);

	// Return if all sampled pixels are going to be out of image area
	if(prev_frame_pixel_i.x < -1 || prev_frame_pixel_i.y < -1 ||
	   prev_frame_pixel_i.x >= w || prev_frame_pixel_i.y >= h
	)
	{
		store3(result_frame, linear_pixel, my_new_color);
		return;
	}

	// Compute the color AABB in the 3x3 neighbourhood and the min/max in a cross pattern around the current pixel
	vec3 minimum_box	= vec3(+C_FLT_MAX);
	vec3 minimum_cross	= vec3(+C_FLT_MAX);
	vec3 maximum_box	= vec3(-C_FLT_MAX);
	vec3 maximum_cross	= vec3(-C_FLT_MAX);
	for(int y = -1; y <= 1; ++y)
	{
		for(int x = -1; x <= 1; ++x)
		{
			ivec2 sample_location = pixel + ivec2(x, y);
			if(sample_location.x >= 0 && sample_location.y >= 0 &&
			   sample_location.x < w && sample_location.y < h
			)
			{
				vec3 sample_color;
				if(x == 0 && y == 0)
					sample_color = my_new_color;
				else
					sample_color = load3<float>(new_frame, sample_location.x + sample_location.y * w);

				sample_color = RGB_to_YCoCg(sample_color);

				if(x == 0 || y == 0)
				{
					minimum_cross = Min(minimum_cross, sample_color);
					maximum_cross = Max(maximum_cross, sample_color);
				}

				minimum_box = Min(minimum_box, sample_color);
				maximum_box = Max(maximum_box, sample_color);
			}
		}
	}

	// Bilinear sampling of previous frame.
	// Note: work-item has already returned if the sampling location is completly out of image
	vec3 prev_color = vec3(0.f, 0.f, 0.f);
	float total_weight = 0;
	const vec2 pixel_fract = prev_frame_pixel_f - vec2(prev_frame_pixel_i);
	const vec2 one_minus_pixel_fract = 1.f - pixel_fract;

	if(prev_frame_pixel_i.y >= 0)
	{
		if(prev_frame_pixel_i.x >= 0)
		{
			float weight = one_minus_pixel_fract.x * one_minus_pixel_fract.y;
			prev_color += weight * load3<float>(prev_frame, prev_frame_pixel_i.y * w + prev_frame_pixel_i.x);
			total_weight += weight;
		}

		if(prev_frame_pixel_i.x < w - 1)
		{
			float weight = pixel_fract.x * one_minus_pixel_fract.y;
			prev_color += weight * load3<float>(prev_frame, prev_frame_pixel_i.y * w + prev_frame_pixel_i.x + 1);
			total_weight += weight;
		}
	}

	if(prev_frame_pixel_i.y < h - 1)
	{
		if(prev_frame_pixel_i.x >= 0)
		{
			float weight = one_minus_pixel_fract.x * pixel_fract.y;
			prev_color += weight * load3<float>(prev_frame, (prev_frame_pixel_i.y + 1) * w + prev_frame_pixel_i.x);
			total_weight += weight;
		}

		if(prev_frame_pixel_i.x < w - 1)
		{
			float weight = pixel_fract.x * pixel_fract.y;
			prev_color += weight * load3<float>(prev_frame, (prev_frame_pixel_i.y + 1) * w + prev_frame_pixel_i.x + 1);
			total_weight += weight;
		}
	}

	if(total_weight > 0)
		prev_color /= total_weight; // Total weight can be less than one on the edges

	vec3 prev_color_ycocg = RGB_to_YCoCg(prev_color);

	// Note: Some references use more complicated methods to move the previous frame color to the YCoCg space AABB
	vec3 minimum = (minimum_box + minimum_cross) / 2.f;
	vec3 maximum = (maximum_box + maximum_cross) / 2.f;
	vec3 prev_color_rgb = YCoCg_to_RGB(Clamp(prev_color_ycocg, minimum, maximum));

	vec3 result_color = TAA_BLEND_ALPHA * my_new_color + (1.f - TAA_BLEND_ALPHA) * prev_color_rgb; // Lerp(prev_color_rgb, my_new_color, TAA_BLEND_ALPHA);
	store3(result_frame, linear_pixel, result_color);
}

extern "C" void run_new_taa(
	dim3 const & grid_size,
	dim3 const & block_size,
	TAAKernelParams const & params,
	const vec2 * K_RESTRICT in_prev_frame_pixel,	// [in]  Previous frame pixel coordinates (after reprojection)
	const float * K_RESTRICT new_frame,				// [in]	 Current frame color buffer
		  float * K_RESTRICT result_frame,			// [out] Antialiased frame color buffer
	const float * K_RESTRICT prev_frame				// [in]  Previous frame color buffer
)
{
	new_taa<<<grid_size, block_size>>>(
		params,
		in_prev_frame_pixel,
		new_frame,
		result_frame,
		prev_frame
	);
}
