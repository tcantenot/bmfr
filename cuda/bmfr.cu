#include "hip/hip_runtime.h"


inline void SyncThreads()
{
	__syncthreads();
}

// Unrolled parallel sum reduction of 256 values
// TODO: unused start_index...
inline void parallel_reduction_sum_256(float * result, volatile float * pr_data_256, const int start_index)
{
	const int id = threadIdx.x;

	if(id < 64)
		pr_data_256[id] += pr_data_256[id + 64] + pr_data_256[id + 128] + pr_data_256[id + 192];
	SyncThreads();

	if(id < 8)
		pr_data_256[id] += pr_data_256[id + 8]  + pr_data_256[id + 16] + pr_data_256[id + 24] +
						   pr_data_256[id + 32] + pr_data_256[id + 40] + pr_data_256[id + 48] + pr_data_256[id + 56];
	SyncThreads();

	if(id == 0)
	{
		*result = pr_data_256[0] + pr_data_256[1] + pr_data_256[2] + pr_data_256[3] +
				  pr_data_256[4] + pr_data_256[5] + pr_data_256[6] + pr_data_256[7];
	}
	SyncThreads();
}

// TODO: replace by Min4
// Unrolled parallel min reduction of 256 values
inline void parallel_reduction_min_256(float * result, volatile float * pr_data_256)
{
	const int id = threadIdx.x;

	if(id < 64)
		pr_data_256[id] = Min(Min(Min(pr_data_256[id], pr_data_256[id + 64]), pr_data_256[id + 128]), pr_data_256[id + 192]);
	SyncThreads();

	if(id < 8)
		pr_data_256[id] = Min(Min(Min(Min(Min(Min(Min(pr_data_256[id], pr_data_256[id + 8]),
			pr_data_256[id + 16]), pr_data_256[id + 24]), pr_data_256[id + 32]), pr_data_256[id + 40]),
			pr_data_256[id + 48]), pr_data_256[id + 56]);
	SyncThreads();

	if(id == 0)
	{
		*result = Min(Min(Min(Min(Min(Min(Min(pr_data_256[0], pr_data_256[1]), pr_data_256[2]),
			pr_data_256[3]), pr_data_256[4]), pr_data_256[5]), pr_data_256[6]), pr_data_256[7]);
	}
	SyncThreads();
}

// TODO: replace by Max4
// Unrolled parallel max reduction of 256 values
inline void parallel_reduction_max_256(float * result, volatile float * pr_data_256)
{
   const int id = threadIdx.x;

	if(id < 64)
		pr_data_256[id] = Max(Max(Max(pr_data_256[id], pr_data_256[id + 64]), pr_data_256[id + 128]), pr_data_256[id + 192]);
	SyncThreads();

	if(id < 8)
		pr_data_256[id] = Max(Max(Max(Max(Max(Max(Max(pr_data_256[id], pr_data_256[id + 8]),
			pr_data_256[id + 16]), pr_data_256[id + 24]), pr_data_256[id + 32]), pr_data_256[id + 40]),
			pr_data_256[id + 48]), pr_data_256[id + 56]);
	SyncThreads();

	if(id == 0)
	{
		*result = Max(Max(Max(Max(Max(Max(Max(pr_data_256[0], pr_data_256[1]), pr_data_256[2]),
			pr_data_256[3]), pr_data_256[4]), pr_data_256[5]), pr_data_256[6]), pr_data_256[7]);
	}
	SyncThreads();
}

// TODO: change these either by macro that take parameters or inline functions
// Helper defines used in IN_ACCESS define
#define BLOCK_EDGE_HALF (BLOCK_EDGE_LENGTH / 2)
#define HORIZONTAL_BLOCKS (WORKSET_WIDTH / BLOCK_EDGE_LENGTH)
#define BLOCK_INDEX_X (group_id % (HORIZONTAL_BLOCKS + 1))
#define BLOCK_INDEX_Y (group_id / (HORIZONTAL_BLOCKS + 1))
#define IN_BLOCK_INDEX (BLOCK_INDEX_Y * (HORIZONTAL_BLOCKS + 1) + BLOCK_INDEX_X)
#define FEATURE_START (feature_buffer * BLOCK_PIXELS)
#define IN_ACCESS (IN_BLOCK_INDEX * buffers * BLOCK_PIXELS + FEATURE_START + sub_vector * 256 + id)
